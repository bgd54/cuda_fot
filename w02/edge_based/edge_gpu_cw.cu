#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <string.h>
#include "graph_helper.hpp"
#include "rms.hpp"
#define TIMER_MACRO
#include "simulation.hpp"
#include "coloring.hpp"

using namespace std;

#define BLOCKSIZE 128

////////////////////////////////////////////////////////////////////////////////
// CPU routines
////////////////////////////////////////////////////////////////////////////////

void addTimers(Simulation &sim){
  #ifdef TIMER_MACRO
  sim.timers.push_back(timer("color"));
  sim.timers.push_back(timer("colorb"));
  sim.timers.push_back(timer("calc_cache"));
  #endif
}

void cache_map_gen(int* enode, int nedge, int* iwillwritethis, int* icachethis,
    Block_coloring& bc){

  for(int i=0;i<nedge;++i) icachethis[i]=-1;

  for(int bIdx=0; bIdx<bc.numblock;++bIdx){
    int start= bIdx*bc.bs;
    int end= std::min((bIdx+1)*bc.bs,nedge);
    std::set<int> needtoCacheforWrite;
    for(int tid=0; tid + start < end; ++tid){
      //kigyujtom a nodeidkat amiket irni fogok szalankent es osszessegeben
      iwillwritethis[start+tid]=enode[2*bc.color_reord[start+tid]+1];
      needtoCacheforWrite.insert(iwillwritethis[start+tid]);
    }

    std::copy(needtoCacheforWrite.begin(),
        needtoCacheforWrite.end(), icachethis+start);
    for(int tid=0;tid+start<end;++tid){
      iwillwritethis[start+tid] = 
        std::find(icachethis+start, icachethis+end, iwillwritethis[start+tid])
        - (icachethis+start);
    }

  }
  /*
  for(int bIdx=0; bIdx<bc.numblock;++bIdx){
    int start= bIdx*bc.bs;
    int end= std::min((bIdx+1)*bc.bs,nedge);
    for(int tid=0; tid + start < end; ++tid){
      printf("bIdx: %3d tid: %3d i: %3d icache: %6d, iwrite: %3d, which: %6d\n", bIdx, tid, start+tid,
         icachethis[start+tid], iwillwritethis[start+tid],
         enode[2*bc.color_reord[start+tid]+1]);
    }
  }
  */

}

////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////
__global__ void ssoln(float* old, const float* val, const int nnode){
  int tid = blockDim.x*blockIdx.x+threadIdx.x;
  if(tid < nnode){
    old[tid]=val[tid];
  }
}

__global__ void iter_calc(const float* old, float* val,const float* eval,
    const int* enode, const int* color_reord, const int nedge,
    const int* color, const int* colornum, const int* blocksInColor,
    int color_start, int* cache_map, int* global_to_cache){

  int tid = threadIdx.x;
  extern  __shared__  float tempval[];


  int bIdx = blocksInColor[blockIdx.x+color_start];
  int reordIdx = tid + bIdx*blockDim.x;

  int iwritethisIdx = -1;
  int iloadThis = -1;

  if(reordIdx<nedge){
    iwritethisIdx = cache_map[reordIdx];
    iloadThis = global_to_cache[reordIdx];
    if(iloadThis != -1) tempval[tid] = val[iloadThis];
  }
  __syncthreads();


  float increment = 0.0f;
  if(reordIdx < nedge){
    increment = 
      eval[color_reord[reordIdx]]*old[enode[color_reord[reordIdx]*2+0]];
  }
  for(int col=0; col<colornum[bIdx];++col){
    if(reordIdx < nedge && col == color[reordIdx]){
      //val[enode[2*color_reord[reordIdx]+1] ] += increment;
      tempval[iwritethisIdx] += increment;
    }
    __syncthreads();
  }
  //cachelt ertekek visszairasa

  if(reordIdx<nedge && iloadThis != -1) val[iloadThis] = tempval[tid];
  
}

///___________________________________________________________________________
int main(int argc, char *argv[]){
  int niter=1000;
  int dx = 1000, dy = 2000;
  bool bidir=false;
  ///////////////////////////////////////////////////////////////////////
  //                            params
  ///////////////////////////////////////////////////////////////////////
  for(int i=1; i < argc; ++i){
    if (!strcmp(argv[i],"-niter")) niter=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-dx")) dx=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-dy")) dy=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-bidir")) bidir=true;
    else {
      fprintf(stderr,"Error: Command-line argument '%s' not recognized.\n",
          argv[i]);
      exit(-1);
    }
  }
  ///////////////////////////////////////////////////////////////////////
  //                            graph gen
  ///////////////////////////////////////////////////////////////////////

  int nnode, nedge;
  int* enode = bidir ? 
    generate_bidirected_graph(dx,dy,nedge,nnode) : 
    generate_graph(dx,dy,nedge,nnode);

  float *node_val, *node_old, *edge_val;
  
  node_val = genDataForNodes(nnode,1);
  edge_val = genDataForNodes(nedge,1);
  
  node_old=(float*)malloc(nnode*sizeof(float));
  ///////////////////////////////////////////////////////////////////////
  //                            timer
  ///////////////////////////////////////////////////////////////////////
  Simulation sim = initSimulation(nedge, nnode);
  addTimers(sim);


  /////////////////////////////////////////////////////////
  //                        coloring
  /////////////////////////////////////////////////////////
  
  printf("start coloring\n");
  TIMER_START(sim.timers[0])
  Block_coloring c = block_coloring(enode,nedge,nnode);
  TIMER_STOP(sim.timers[0])
  printf("start coloring blocks\n");
  TIMER_START(sim.timers[1])
  Coloring bc = c.color_blocks(enode,nedge);
  TIMER_STOP(sim.timers[1])
  printf("ready\n");
  printf("calculate cacheable data\n");
  TIMER_START(sim.timers[2])
  int* iwillwritethis, *icachethis;
  iwillwritethis= (int*) malloc(nedge*sizeof(int));
  icachethis    = (int*) malloc(nedge*sizeof(int));
  
  cache_map_gen(enode, nedge, iwillwritethis, icachethis, c); 
  TIMER_STOP(sim.timers[2])

  /////////////////////////////////////
  //          Device pointers
  /////////////////////////////////////
  printf("coloring ready, allocate arrays in device memory\n");
  int *enode_d, *color_reord_d, *colornum_d, *color_d;
  float *node_val_d,*node_old_d,*edge_val_d;
  int *block_reord_d;
  int *iwillwritethis_d, *icachethis_d;

  checkCudaErrors( hipMalloc((void**)&enode_d, 2*nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&color_reord_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&color_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&colornum_d, c.numblock*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&edge_val_d, nedge*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&node_old_d, nnode*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&node_val_d, nnode*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&block_reord_d, c.numblock*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&iwillwritethis_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&icachethis_d, nedge*sizeof(int)) );
  
  checkCudaErrors( hipMemcpy(enode_d, enode, 2*nedge*sizeof(int),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(color_reord_d, c.color_reord,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(color_d, c.reordcolor,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(colornum_d, c.colornum, c.numblock*sizeof(int),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(edge_val_d, edge_val, nedge*sizeof(float),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(node_val_d, node_val, nnode*sizeof(float),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(block_reord_d, bc.color_reord,
                               c.numblock*sizeof(int),
                               hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(iwillwritethis_d, iwillwritethis,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(icachethis_d, icachethis,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );

  ///////////////////////////////////////////////////////////
  //                      Start
  ///////////////////////////////////////////////////////////
  printf("start edge based on CPU niter: %d, nnode:%d, nedge:%d, numblock: %d\n",niter,
     nnode,nedge, c.numblock);
  //   timer
  sim.start();
  //______________________________main_loop_____________________________
  for(int i=0;i<=niter;++i){
    //save old
    sim.kernels[0].timerStart();
    ssoln<<<(nnode-1)/BLOCKSIZE+1,BLOCKSIZE>>>(node_old_d,node_val_d, nnode);
    checkCudaErrors( hipDeviceSynchronize() );
    sim.kernels[0].timerStop();


    //calc next step
    for(int col=0; col<bc.colornum;col++){ 
      int start = col==0?0:bc.color_offsets[col-1]; 
      int len = bc.color_offsets[col]-start;
      sim.kernels[1].timerStart();
      iter_calc<<<len,BLOCKSIZE,BLOCKSIZE*sizeof(float)>>>(node_old_d,
          node_val_d, edge_val_d, enode_d, color_reord_d, nedge, color_d,
          colornum_d, block_reord_d, start, iwillwritethis_d, icachethis_d);
      checkCudaErrors( hipDeviceSynchronize() );
      sim.kernels[1].timerStop();
    }

    // rms
    if(i%100==0){
      sim.kernels[2].timerStart();
      checkCudaErrors( hipMemcpy(node_val, node_val_d, nnode*sizeof(float),
                              hipMemcpyDeviceToHost) );
      checkCudaErrors( hipMemcpy(node_old, node_old_d, nnode*sizeof(float),
                              hipMemcpyDeviceToHost) );
      rms_calc(node_val,node_old,nnode,i);
      sim.kernels[2].timerStop();

    }

  }
  //____________________________end main loop___________________________
  //    timer
  sim.stop();

  sim.printTiming();

  
  //free
  free(enode);
  free(node_old);
  free(node_val);
  free(edge_val);
  free(iwillwritethis);
  free(icachethis);
  //cuda freee
  checkCudaErrors( hipFree(enode_d) );
  checkCudaErrors( hipFree(color_reord_d) );
  checkCudaErrors( hipFree(edge_val_d) );
  checkCudaErrors( hipFree(node_old_d) );
  checkCudaErrors( hipFree(node_val_d) );
  checkCudaErrors( hipFree(color_d) );
  checkCudaErrors( hipFree(colornum_d) );
  checkCudaErrors( hipFree(block_reord_d) );
  checkCudaErrors( hipFree(iwillwritethis_d) );
  checkCudaErrors( hipFree(icachethis_d) );
  
  return 0;
}
