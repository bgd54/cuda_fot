#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <string.h>
#include "graph_helper.hpp"
#include "rms.hpp"
#define TIMER_MACRO
#include "simulation.hpp"
#include "coloring.hpp"

using namespace std;

#define BLOCKSIZE 128

void addTimers(Simulation &sim){
  #ifdef TIMER_MACRO
  sim.timers.push_back(timer("color"));
  sim.timers.push_back(timer("colorb"));
  sim.timers.push_back(timer("cache_calc"));
  #endif
}

////////////////////////////////////////////////////////////////////////////////
// CPU routines
////////////////////////////////////////////////////////////////////////////////

void cache_map_gen(int* enode, int nedge, int* iwillwritethis, int* icachethis,
    Block_coloring& bc){

  for(int i=0;i<nedge;++i) icachethis[i]=-1;

  for(int bIdx=0; bIdx<bc.numblock;++bIdx){
    int start= bIdx*bc.bs;
    int end= std::min((bIdx+1)*bc.bs,nedge);
    std::set<int> needtoCacheforWrite;
    for(int tid=0; tid + start < end; ++tid){
      //kigyujtom a nodeidkat amiket irni fogok szalankent es osszessegeben
      iwillwritethis[start+tid]=enode[2*bc.color_reord[start+tid]+1];
      needtoCacheforWrite.insert(iwillwritethis[start+tid]);
    }

    std::copy(needtoCacheforWrite.begin(),
        needtoCacheforWrite.end(), icachethis+start);
    for(int tid=0;tid+start<end;++tid){
      iwillwritethis[start+tid] = 
        std::find(icachethis+start, icachethis+end, iwillwritethis[start+tid])
        - (icachethis+start);
    }

  }
  /*
  for(int bIdx=0; bIdx<bc.numblock;++bIdx){
    int start= bIdx*bc.bs;
    int end= std::min((bIdx+1)*bc.bs,nedge);
    for(int tid=0; tid + start < end; ++tid){
      printf("bIdx: %3d tid: %3d i: %3d icache: %6d, iwrite: %3d, which: %6d\n", bIdx, tid, start+tid,
         icachethis[start+tid], iwillwritethis[start+tid],
         enode[2*bc.color_reord[start+tid]+1]);
    }
  }
  */

}
void cache_map_read(int* enode, int nedge, int* ireadold, int* icachethis_old,
    int* eval_ind, int* eval_cache,Block_coloring& bc){

  for(int i=0;i<nedge;++i){ 
    icachethis_old[i]=-1;
    eval_cache[i]=-1;
  }


  for(int bIdx=0; bIdx<bc.numblock;++bIdx){
    int start= bIdx*bc.bs;
    int end= std::min((bIdx+1)*bc.bs,nedge);
    std::set<int> needtoCacheforOld;
    std::set<int> needtoCacheforEval;
    for(int tid=0; tid + start < end; ++tid){
      //kigyujtom a nodeidkat amiket irni fogok szalankent es osszessegeben
      ireadold[start+tid] = enode[2*bc.color_reord[start+tid]+0];
      eval_ind[start+tid] = bc.color_reord[start+tid];
      needtoCacheforOld.insert(ireadold[start+tid]);
      needtoCacheforEval.insert(eval_ind[start+tid]);
    }

    std::copy(needtoCacheforOld.begin(),
        needtoCacheforOld.end(), icachethis_old+start);
    std::copy(needtoCacheforEval.begin(),
        needtoCacheforEval.end(), eval_cache+start);
    for(int tid=0;tid+start<end;++tid){
      ireadold[start+tid] = 
        std::find(icachethis_old+start, icachethis_old+end, ireadold[start+tid])
        - (icachethis_old+start);
      eval_ind[start+tid] = 
        std::find(eval_cache+start, eval_cache+end, eval_ind[start+tid])
        - (eval_cache+start);
    }

  }
}

////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////
__global__ void ssoln(float* old, const float* val, const int nnode, const int node_dim){
  int tid = blockDim.x*blockIdx.x+threadIdx.x;
  if(tid < nnode*node_dim){
    old[tid]=val[tid];
  }
}

__global__ void iter_calc(const float* old, float* val,const float* eval,
    const int* enode, const int* color_reord, const int nedge,
    const int* color, const int* colornum, const int* blocksInColor,
    int color_start, int* cache_map, int* global_to_cache, int * cache_old_map,
    int* global_cache_old_map, int* cache_eval_map, int* global_cache_eval_map, const int node_dim){

  int tid = threadIdx.x;
  extern  __shared__  float tempval[];


  int bIdx = blocksInColor[blockIdx.x+color_start];
  int reordIdx = tid + bIdx*blockDim.x;

  int iwritethisIdx = -1;
  int iloadThis = -1;
  int ireadOld = -1;
  int ireadeval = -1;
  int icacheOld = -1;
  int icahceEval = -1;

  if(reordIdx<nedge){
    iwritethisIdx = cache_map[reordIdx];
    iloadThis = global_to_cache[reordIdx];

    ireadOld = cache_old_map[reordIdx];
    icacheOld = global_cache_old_map[reordIdx];
    
    ireadeval = cache_eval_map[reordIdx];
    icahceEval = global_cache_eval_map[reordIdx];
    
    if(iloadThis != -1){
      for(int dim=0; dim<node_dim;dim++){ 
        tempval[tid*node_dim+dim] = val[iloadThis*node_dim+dim];
      }
    }
    if(icacheOld != -1){
      for(int dim=0; dim<node_dim;dim++){ 
        tempval[tid*node_dim+blockDim.x*node_dim+dim] =
          old[icacheOld*node_dim+dim];
      }
    }
    if(icahceEval != -1) tempval[tid+2*blockDim.x*node_dim] = eval[icahceEval];
  }
  __syncthreads();


  float* increment = new float[node_dim];
  if(reordIdx < nedge){

    for(int dim=0; dim<node_dim;dim++){ 
      increment[dim] = tempval[ireadeval+2*blockDim.x*node_dim] *
        tempval[(ireadOld+blockDim.x)*node_dim+dim];
    }

      //eval[color_reord[reordIdx]]*old[enode[color_reord[reordIdx]*2+0]];
  }
  for(int col=0; col<colornum[bIdx];++col){
    if(reordIdx < nedge && col == color[reordIdx]){
      for(int dim=0; dim<node_dim;dim++){ 
        tempval[iwritethisIdx*node_dim+dim]+= increment[dim];
      }

      //val[enode[2*color_reord[reordIdx]+1] ] += increment;
    }
    __syncthreads();
  }
  //cachelt ertekek visszairasa

  if(reordIdx<nedge && iloadThis != -1){
    for(int dim=0; dim<node_dim;dim++){ 
      val[iloadThis*node_dim+dim] = tempval[tid*node_dim+dim];
    }
  }
}

///___________________________________________________________________________
int main(int argc, char *argv[]){
  int niter=1000;
  int dx = 1000, dy = 2000;
  bool bidir=false;
  int node_dim = 1, edge_dim = 1;
  ///////////////////////////////////////////////////////////////////////
  //                            params
  ///////////////////////////////////////////////////////////////////////
  for(int i=1; i < argc; ++i){
    if (!strcmp(argv[i],"-niter")) niter=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-dx")) dx=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-dy")) dy=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-bidir")) bidir=true;
    else if (!strcmp(argv[i],"-ndim")) node_dim=atoi(argv[++i]);
    else {
      fprintf(stderr,"Error: Command-line argument '%s' not recognized.\n",
          argv[i]);
      exit(-1);
    }
  }
  ///////////////////////////////////////////////////////////////////////
  //                            graph gen
  ///////////////////////////////////////////////////////////////////////

  int nnode, nedge;
  int* enode = bidir ? 
    generate_bidirected_graph(dx,dy,nedge,nnode) : 
    generate_graph(dx,dy,nedge,nnode);

  float *node_val, *node_old, *edge_val;
  
  node_val = genDataForNodes(nnode,node_dim);
  edge_val = genDataForNodes(nedge,edge_dim);
  
  node_old=(float*)malloc(nnode*node_dim*sizeof(float));
  ///////////////////////////////////////////////////////////////////////
  //                            timer
  ///////////////////////////////////////////////////////////////////////
  Simulation sim = initSimulation(nedge, nnode, node_dim);
  addTimers(sim);



  /////////////////////////////////////////////////////////
  //                        coloring
  /////////////////////////////////////////////////////////
  
  printf("start coloring\n");
  TIMER_START(sim.timers[0])
  
  Block_coloring c = block_coloring(enode,nedge);
  TIMER_STOP(sim.timers[0])
  printf("start coloring blocks\n");
  TIMER_START(sim.timers[1])
  Coloring bc = c.color_blocks(enode,nedge);
  TIMER_STOP(sim.timers[1])
  
  printf("ready\n");
  printf("calculate cacheable data\n");
  TIMER_START(sim.timers[2])
  int* iwillwritethis, *icachethis;
  int* ireadold, *icachethis_old;
  int* ireadeval, *icachethis_eval;
  iwillwritethis      = (int*) malloc(nedge*sizeof(int));
  icachethis          = (int*) malloc(nedge*sizeof(int));
  ireadold            = (int*) malloc(nedge*sizeof(int));
  icachethis_old      = (int*) malloc(nedge*sizeof(int));
  ireadeval           = (int*) malloc(nedge*sizeof(int));
  icachethis_eval     = (int*) malloc(nedge*sizeof(int));
  
  cache_map_gen(enode, nedge, iwillwritethis, icachethis, c); 
  cache_map_read(enode, nedge, ireadold, icachethis_old,
     ireadeval, icachethis_eval, c); 
  TIMER_STOP(sim.timers[2])

  /////////////////////////////////////
  //          Device pointers
  /////////////////////////////////////
  printf("coloring ready, allocate arrays in device memory\n");
  int *enode_d, *color_reord_d, *colornum_d, *color_d;
  float *node_val_d,*node_old_d,*edge_val_d;
  int *block_reord_d;
  int *iwillwritethis_d, *icachethis_d;

  checkCudaErrors( hipMalloc((void**)&enode_d, 2*nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&color_reord_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&color_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&colornum_d, c.numblock*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&edge_val_d, nedge*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&node_old_d, nnode*node_dim*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&node_val_d, nnode*node_dim*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&block_reord_d, c.numblock*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&iwillwritethis_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&icachethis_d, nedge*sizeof(int)) );
  
  checkCudaErrors( hipMemcpy(enode_d, enode, 2*nedge*sizeof(int),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(color_reord_d, c.color_reord,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(color_d, c.reordcolor,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(colornum_d, c.colornum, c.numblock*sizeof(int),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(edge_val_d, edge_val, nedge*sizeof(float),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(node_val_d, node_val, nnode*node_dim*sizeof(float),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(block_reord_d, bc.color_reord,
                               c.numblock*sizeof(int),
                               hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(iwillwritethis_d, iwillwritethis,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(icachethis_d, icachethis,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );

  //////////////////// device pointers for read caches //////////////////////

  int *ireadold_d, *icachethis_old_d;
  int *ireadeval_d, *icachethis_eval_d;
  checkCudaErrors( hipMalloc((void**)&ireadold_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&icachethis_old_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&ireadeval_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&icachethis_eval_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMemcpy(ireadold_d, ireadold,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(icachethis_old_d, icachethis_old,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(ireadeval_d, ireadeval,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(icachethis_eval_d, icachethis_eval,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );

  ///////////////////////////////////////////////////////////
  //                      Start
  ///////////////////////////////////////////////////////////
  printf("start edge based on CPU niter: %d, nnode:%d, nedge:%d, numblock: %d\n",niter,
     nnode,nedge, c.numblock);
  //   timer
  sim.start();
  //______________________________main_loop_____________________________
  for(int i=0;i<=niter;++i){
    //save old
    sim.kernels[0].timerStart();
    ssoln<<<(nnode*node_dim-1)/BLOCKSIZE+1,BLOCKSIZE>>>(node_old_d,node_val_d, nnode, node_dim);
    checkCudaErrors( hipDeviceSynchronize() );
    sim.kernels[0].timerStop();

    //calc next step
    for(int col=0; col<bc.colornum;col++){ 
      int start = col==0?0:bc.color_offsets[col-1]; 
      int len = bc.color_offsets[col]-start;
      sim.kernels[1].timerStart();
      iter_calc<<<len,BLOCKSIZE,3*BLOCKSIZE*node_dim*sizeof(float)>>>(node_old_d, //2*nodedim+1*BS?
          node_val_d, edge_val_d, enode_d, color_reord_d, nedge, color_d,
          colornum_d, block_reord_d, start, iwillwritethis_d, icachethis_d,
          ireadold_d, icachethis_old_d, ireadeval_d, icachethis_eval_d, node_dim);
      checkCudaErrors( hipDeviceSynchronize() );
      sim.kernels[1].timerStop();
    }

    // rms
    if(i%100==0){
      sim.kernels[2].timerStart();
      checkCudaErrors( hipMemcpy(node_val, node_val_d, nnode*node_dim*sizeof(float),
                              hipMemcpyDeviceToHost) );
      checkCudaErrors( hipMemcpy(node_old, node_old_d, nnode*node_dim*sizeof(float),
                              hipMemcpyDeviceToHost) );
      rms_calc(node_val,node_old,nnode,i,node_dim);
      sim.kernels[2].timerStop();
    }

  }
  //____________________________end main loop___________________________
  //    timer
  sim.stop();

  sim.printTiming();

  //free
  free(enode);
  free(node_old);
  free(node_val);
  free(edge_val);
  free(iwillwritethis);
  free(icachethis);
  free(ireadold);
  free(icachethis_old);
  free(ireadeval);
  free(icachethis_eval);
  //cuda freee
  checkCudaErrors( hipFree(enode_d) );
  checkCudaErrors( hipFree(color_reord_d) );
  checkCudaErrors( hipFree(edge_val_d) );
  checkCudaErrors( hipFree(node_old_d) );
  checkCudaErrors( hipFree(node_val_d) );
  checkCudaErrors( hipFree(color_d) );
  checkCudaErrors( hipFree(colornum_d) );
  checkCudaErrors( hipFree(block_reord_d) );
  checkCudaErrors( hipFree(iwillwritethis_d) );
  checkCudaErrors( hipFree(icachethis_d) );
  checkCudaErrors( hipFree(icachethis_old_d) );
  checkCudaErrors( hipFree(ireadold_d) );
  checkCudaErrors( hipFree(ireadeval_d) );
  checkCudaErrors( hipFree(icachethis_eval_d) );
  
  
  
  return 0;
}
