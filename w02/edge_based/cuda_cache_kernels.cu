#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "helper_string.h"
#include "kernels.hpp"
#include "ssol_cudakernel.cu"

__global__ void iter_calc( const float* __restrict__  old, 
    float* __restrict__ val, const int node_dim ,const float* __restrict__ eval,
    const int nedge, const int* __restrict__ enode, 
    const int* __restrict__ color_reord, const int* __restrict__ threadcolors,
    const int* __restrict__ colornum, const int* __restrict__ blocksInColor,
    const int color_start, const int* __restrict__ global_to_cache, 
    const int* __restrict__ cacheOffsets,
    const int* __restrict__ global_read_to_cache,
    const int* __restrict__ cacheReadOffsets,
    const MY_IDX_TYPE* __restrict__ writeC,
    const MY_IDX_TYPE* __restrict__ readC ){

  int tid = threadIdx.x;
  extern  __shared__  float shared[];


  int bIdx = blocksInColor[blockIdx.x+color_start];
  int reordIdx = tid + bIdx*blockDim.x;

  int iwritethisIdx = -1;
  int ireadthisIdx = -1;

  if(reordIdx<nedge){
    iwritethisIdx = writeC[reordIdx];
    ireadthisIdx = readC[reordIdx];
  }
  //calc cache params
  int cache_offset = bIdx == 0? 0:cacheOffsets[bIdx-1]; 
  int cache_size = cacheOffsets[bIdx] - cache_offset;
  int read_cache_offset = bIdx == 0? 0:cacheReadOffsets[bIdx-1]; 
  int read_cache_size = cacheReadOffsets[bIdx] - read_cache_offset;
  //set pointers to cache
  float* valC = shared;
  float* oldC = shared + cache_size;
  //CACHE IN
  for (int i = 0; i < cache_size; i += blockDim.x) {
    if (i + tid < cache_size) {
      for(int dim=0; dim<node_dim; dim++)
        valC[(i + tid)*node_dim+dim] =
            val[global_to_cache[cache_offset + i + tid]*node_dim+dim];
    }
  }
  for (int i = 0; i < read_cache_size; i += blockDim.x) {
    if (i + tid < read_cache_size) {
      for(int dim=0; dim<node_dim; dim++)
        oldC[(i + tid)*node_dim+dim] = old[
              global_read_to_cache[read_cache_offset + i + tid]*node_dim+dim];
    }
  }
 
  __syncthreads();

  //CALC INCREMENT
  float increment[MAX_NODE_DIM];
  int edgeIdx=0;
  int mycolor=-1;
  if(reordIdx < nedge){
    edgeIdx=color_reord[reordIdx];
    mycolor = threadcolors[reordIdx];
    for(int dim=0; dim<node_dim;dim++){ 
      increment[dim] = eval[edgeIdx]*oldC[ireadthisIdx*node_dim+dim];
    }
  }

  //CALC VAL
  for(int col=0; col<colornum[bIdx];++col){
    if(reordIdx < nedge && col == mycolor){
      for(int dim=0; dim<node_dim;dim++){ 
        valC[iwritethisIdx*node_dim+dim]+= increment[dim];
      }
    }
    __syncthreads();
  }

  //CACHE BACK
  for (int i = 0; i < cache_size; i += blockDim.x) {
    if (i + tid < cache_size) {
      for(int dim=0; dim<node_dim; dim++)
        val[global_to_cache[cache_offset + i + tid]*node_dim+dim] = 
          valC[(i + tid)*node_dim+dim];
    }
  }
 
}


void iter_calc(const int nedge, const int nnode, const int node_dim,
   const Block_coloring& bc, const Coloring& c, const arg& arg_enode,
   const arg& arg_edge_val, arg& arg_node_val, const arg& arg_node_old,
   cacheMap& cm, Kernel& timer){

  int* enode_d = (int*) arg_enode.data_d;
  int* color_reord_d = (int *) bc.arg_color_reord.data_d;
  int* color_d = (int *) bc.arg_reordcolor.data_d;
  int* colornum_d = (int *) bc.arg_colornum.data_d;
  int* block_reord_d = (int *) c.arg_color_reord.data_d;
  float * node_val_d = (float*) arg_node_val.data_d;
  float * node_old_d = (float*) arg_node_old.data_d;
  float * edge_val_d = (float*) arg_edge_val.data_d;
  
  //calc next step
  for(int col=0; col<c.colornum;col++){ 
    int start = col==0?0:c.color_offsets[col-1]; 
    int len = c.color_offsets[col]-start;
    timer.timerStart();
    //TODO shared memory calc.. 4*->worst case
    iter_calc<<<len,BLOCKSIZE,4*BLOCKSIZE*node_dim*sizeof(float)>>>(
        node_old_d, node_val_d, node_dim, edge_val_d, nedge, enode_d,
        color_reord_d, color_d, colornum_d, block_reord_d, start,
        cm.globalToCacheMap_d, cm.blockOffsets_d,
        cm.globalReadToCacheMap_d, cm.blockReadOffsets_d,
        cm.writeC_d, cm.readC_d);
    checkCudaErrors( hipDeviceSynchronize() );
    timer.timerStop();
  }

}
