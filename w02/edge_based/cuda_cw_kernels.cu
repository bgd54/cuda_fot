#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "helper_string.h"
#include "kernels.hpp"
#include "ssol_cudakernel.cu"

template <int node_dim>
__global__ void iter_calc( const float* __restrict__  old, 
    float* __restrict__ val, const int nnode,
    const float* __restrict__ eval, const int nedge,
    const int* __restrict__ enode,
    const int* __restrict__ threadcolors, const int* __restrict__ colornum,
    const int color_start,
    const int* __restrict__ global_to_cache,
    const int* __restrict__ cacheOffsets,
    const MY_IDX_TYPE* __restrict__ writeC){

  int tid = threadIdx.x;
  extern  __shared__  float shared[];


  int bIdx = blockIdx.x+color_start;
  int reordIdx = tid + bIdx*blockDim.x;

  int iwritethisIdx = -1;

  if(reordIdx<nedge){
    iwritethisIdx = writeC[reordIdx];
  }
  //calc cache params
  int cache_offset = bIdx == 0? 0:cacheOffsets[bIdx-1]; 
  int cache_size = cacheOffsets[bIdx] - cache_offset;
  //set pointers to cache
  float* valC = shared;
  //CACHE IN
  for (int i = 0; i < cache_size; i += blockDim.x) {
    if (i + tid < cache_size) {
      for(int dim=0; dim<node_dim; dim++){
        //Same comment here as for cuda_cache kernels
    #ifdef USE_SOA
      int nodeind = global_to_cache[cache_offset + i + tid]+nnode*dim;
      int cacheind = (i+tid)+cache_size*dim;
    #else
      int nodeind = global_to_cache[cache_offset + i + tid]*node_dim+dim;
      int cacheind = (i+tid)*node_dim + dim;
    #endif
        valC[cacheind] = val[nodeind];
      }
    }
  }
 
  __syncthreads();

  //CALC INCREMENT
  float increment[node_dim];
  int mycolor=-1;
  if(reordIdx < nedge){
    mycolor = threadcolors[reordIdx];
    for(int dim=0; dim<node_dim;dim++){ 
    #ifdef USE_SOA
      int nodeind = enode[reordIdx*2+0] + nnode * dim;
    #else
      int nodeind = enode[reordIdx*2+0]*node_dim + dim;
    #endif
      increment[dim] = eval[reordIdx]*old[nodeind];
    }
  }

  //CALC VAL
  for(int col=0; col<colornum[bIdx];++col){
    if(reordIdx < nedge && col == mycolor){
      for(int dim=0; dim<node_dim;dim++){ 
        //val[enode[2*reordIdx+1]*node_dim+dim] += increment[dim];
      #ifdef USE_SOA
        valC[iwritethisIdx+cache_size*dim] += increment[dim];
      #else
        valC[iwritethisIdx*node_dim+dim] += increment[dim];
      #endif
      }
    }
    __syncthreads();
  }
  //Increment instead of preload? Perhaps an additional version.
  //CACHE BACK
  for (int i = 0; i < cache_size; i += blockDim.x) {
    if (i + tid < cache_size) {
      for(int dim=0; dim<node_dim; dim++){
      #ifdef USE_SOA
        int nodeind = global_to_cache[cache_offset + i + tid]+nnode*dim;
        int cacheind = (i+tid)+cache_size*dim;
      #else
        int nodeind = global_to_cache[cache_offset + i + tid]*node_dim+dim;
        int cacheind = (i+tid)*node_dim + dim;
      #endif
        val[nodeind] = valC[cacheind];
      }
    }
  }
 
}


void iter_calc(const int nedge, const int nnode, const int node_dim,
   const Block_coloring& bc, const Coloring& c, const arg& arg_enode,
   const arg& arg_edge_val, arg& arg_node_val, const arg& arg_node_old,
   cacheMap& cm, Kernel& timer){

  int* enode_d = (int*) arg_enode.data_d;
  int* color_d = (int *) bc.arg_reordcolor.data_d;
  int* colornum_d = (int *) bc.arg_colornum.data_d;
  float * node_val_d = (float*) arg_node_val.data_d;
  float * node_old_d = (float*) arg_node_old.data_d;
  float * edge_val_d = (float*) arg_edge_val.data_d;
  
  //calc next step
  for(int col=0; col<c.colornum;col++){ 
    int start = col==0?0:c.color_offsets[col-1]; 
    int len = c.color_offsets[col]-start;
    timer.timerStart();
    //TODO shared memory calc.. 4*->worst case
    iter_calc<NODE_DIM><<<len,BLOCKSIZE,cm.maxc*node_dim*sizeof(float)>>>(
        node_old_d, node_val_d, nnode, edge_val_d, nedge, enode_d,
        color_d, colornum_d, start, cm.globalToCacheMap_d,
        cm.blockOffsets_d, cm.writeC_d);
    checkCudaErrors( hipDeviceSynchronize() );
    timer.timerStop();
  }

}
