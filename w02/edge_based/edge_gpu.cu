#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <string.h>
#include "graph_helper.hpp"
#include "rms.hpp"
#define TIMER_MACRO
#include "simulation.hpp"
#include "coloring.hpp"

using namespace std;

void addTimers(Simulation &sim){
  #ifdef TIMER_MACRO
  sim.timers.push_back(timer("color"));
  #endif
}

#define BLOCKSIZE 1024
////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////
__global__ void ssoln(float* old, const float* val, const int nnode){
  int tid = blockDim.x*blockIdx.x+threadIdx.x;
  if(tid < nnode){
    old[tid]=val[tid];
  }
}

__global__ void iter_calc(const float* old, float* val,const float* eval,
    const int* enode, const int* color_reord, const int offset, 
    const int color_size, const int nedge){

  int tid = blockDim.x*blockIdx.x+threadIdx.x;
  int reordIdx = tid + offset;
  if(reordIdx<nedge && tid < color_size){
    val[enode[2*color_reord[reordIdx]+1]] +=
      eval[color_reord[reordIdx]]*old[enode[color_reord[reordIdx]*2+0]];
  }
}



int main(int argc, char *argv[]){
  int niter=1000;
  int dx = 1000, dy = 2000;
  bool bidir=false;
  ///////////////////////////////////////////////////////////////////////
  //                            params
  ///////////////////////////////////////////////////////////////////////
  for(int i=1; i < argc; ++i){
    if (!strcmp(argv[i],"-niter")) niter=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-dx")) dx=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-dy")) dy=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-bidir")) bidir=true;
    else {
      fprintf(stderr,"Error: Command-line argument '%s' not recognized.\n",
          argv[i]);
      exit(-1);
    }
  }
  ///////////////////////////////////////////////////////////////////////
  //                            graph gen
  ///////////////////////////////////////////////////////////////////////

  int nnode,nedge;
  int* enode = bidir ? 
    generate_bidirected_graph(dx,dy,nedge,nnode) : 
    generate_graph(dx,dy,nedge,nnode);

  float* node_val,*node_old, *edge_val;
  
  node_val=genDataForNodes(nnode,node_dim);
  edge_val=genDataForNodes(nedge,edge_dim);
  
  node_old=(float*)malloc(nnode*node_dim*sizeof(float));
  ///////////////////////////////////////////////////////////////////////
  //                            timer
  ///////////////////////////////////////////////////////////////////////
  Simulation sim = initSimulation(nedge, nnode);
  addTimers(sim);


  /////////////////////////////////////////////////////////
  //                        coloring
  /////////////////////////////////////////////////////////
  
  printf("start coloring\n");
  TIMER_START(sim.timers[0])
  Coloring c = global_coloring(enode,nedge);
  TIMER_STOP(sim.timers[0])
  printf("coloring ready, allocate arrays in device memory\n");
  /////////////////////////////////////
  //          Device pointers
  /////////////////////////////////////
  int *enode_d, *color_reord_d;
  float *node_val_d,*node_old_d,*edge_val_d;
  
  checkCudaErrors( hipMalloc((void**)&enode_d, 2*nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&color_reord_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&edge_val_d, nedge*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&node_old_d, nnode*node_dim*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&node_val_d, nnode*node_dim*sizeof(float)) );

  checkCudaErrors( hipMemcpy(enode_d, enode, 2*nedge*sizeof(int),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(color_reord_d, c.color_reord,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(edge_val_d, edge_val, nedge*sizeof(float),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(node_val_d, node_val, nnode*node_dim*sizeof(float),
                              hipMemcpyHostToDevice) );

  ///////////////////////////////////////////////////////////
  //                      Start
  ///////////////////////////////////////////////////////////
  printf("start edge based on CPU niter: %d, nnode:%d, nedge:%d, colornum: %d\n",niter,
     nnode,nedge, c.colornum);

  //   timer
  sim.start();
  //______________________________main_loop_____________________________
  for(int i=0;i<=niter;++i){
    //save old
    sim.kernels[0].timerStart();
    ssoln<<<(nnode-1)/BLOCKSIZE+1,BLOCKSIZE>>>(node_old_d,node_val_d, nnode);
    checkCudaErrors( hipDeviceSynchronize() );
    sim.kernels[0].timerStop();


    //calc next step
    for(int col=0; col<c.colornum;col++){ 
      int color_offset = col==0 ? 0 : c.color_offsets[col-1];
      int color_size = c.color_offsets[col] - color_offset;
      sim.kernels[1].timerStart();
      iter_calc<<<(color_size-1)/BLOCKSIZE+1,BLOCKSIZE>>>(node_old_d, 
          node_val_d, edge_val_d, enode_d, color_reord_d, color_offset,
          color_size, nedge);
      checkCudaErrors( hipDeviceSynchronize() );
      sim.kernels[1].timerStop();
    }

    // rms
    if(i%100==0){
      sim.kernels[2].timerStart();
      checkCudaErrors( hipMemcpy(node_val, node_val_d, nnode*node_dim*sizeof(float),
                              hipMemcpyDeviceToHost) );
      checkCudaErrors( hipMemcpy(node_old, node_old_d, nnode*node_dim*sizeof(float),
                              hipMemcpyDeviceToHost) );
      rms_calc(node_val,node_old,nnode,i);
      sim.kernels[2].timerStop();
    }

  }
  //____________________________end main loop___________________________
  //    timer
  sim.stop();

  sim.printTiming();

  //free
  free(enode);
  free(node_old);
  free(node_val);
  free(edge_val);

  //cuda freee
  checkCudaErrors( hipFree(enode_d) );
  checkCudaErrors( hipFree(color_reord_d) );
  checkCudaErrors( hipFree(edge_val_d) );
  checkCudaErrors( hipFree(node_old_d) );
  checkCudaErrors( hipFree(node_val_d) );
  return 0;
}
