#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <string.h>
#include "graph_helper.hpp"
#include "rms.hpp"
#define TIMER_MACRO
#include "timer.hpp"
#include "coloring.hpp"

using namespace std;

#define BLOCKSIZE 128


////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////
__global__ void ssoln(float* old, const float* val, const int nnode){
  int tid = blockDim.x*blockIdx.x+threadIdx.x;
  if(tid < nnode){
    old[tid]=val[tid];
  }
}

__global__ void iter_calc(const float* old, float* val,const float* eval,
    const int* enode, const int* color_reord, const int nedge,
    const int* color, const int* colornum, const int* blocksInColor,
    int color_start){

  int tid = threadIdx.x;
  
  int bIdx = blocksInColor[blockIdx.x+color_start];
  int reordIdx = tid + bIdx*blockDim.x;
  float increment = 0.0f;
  if(reordIdx < nedge){
    increment = 
      eval[color_reord[reordIdx]]*old[enode[color_reord[reordIdx]*2+0]];
  }
  for(int col=0; col<colornum[bIdx];++col){
    if(reordIdx < nedge && col == color[reordIdx]){
      val[enode[2*color_reord[reordIdx]+1] ] += increment;
    }
    __syncthreads();
  }
  //cachelt ertekek visszairasa

}

///___________________________________________________________________________
int main(int argc, char *argv[]){
  int niter=1000;
  int dx = 1000, dy = 2000;
  ///////////////////////////////////////////////////////////////////////
  //                            params
  ///////////////////////////////////////////////////////////////////////
  for(int i=1; i < argc; ++i){
    if (!strcmp(argv[i],"-niter")) niter=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-dx")) dx=atoi(argv[++i]);
    else if (!strcmp(argv[i],"-dy")) dy=atoi(argv[++i]);
    else {
      fprintf(stderr,"Error: Command-line argument '%s' not recognized.\n",
          argv[i]);
      exit(-1);
    }
  }
  ///////////////////////////////////////////////////////////////////////
  //                            graph gen
  ///////////////////////////////////////////////////////////////////////

  int nnode, nedge;
  int* enode = generate_graph(dx, dy, nedge, nnode);

  float *node_val, *node_old, *edge_val;
  
  node_val = genDataForNodes(nnode,1);
  edge_val = genDataForNodes(nedge,1);
  
  node_old=(float*)malloc(nnode*sizeof(float));
  ///////////////////////////////////////////////////////////////////////
  //                            timer
  ///////////////////////////////////////////////////////////////////////
#ifdef TIMER_MACRO
  timer total("total"), ssol("ssol"), iter("iter"), rms("rms"), 
        color("color1"), color2("colorb"); //TODO Attila h oldja ezt meg
#endif

  /////////////////////////////////////////////////////////
  //                        coloring
  /////////////////////////////////////////////////////////
  
  printf("start coloring\n");
  TIMER_START(color)
  Block_coloring c = block_coloring(enode,nedge,nnode);
  TIMER_STOP(color)
  printf("start coloring blocks\n");
  TIMER_START(color2)
  Coloring bc = c.color_blocks(enode,nedge);
  TIMER_STOP(color2)
  printf("coloring ready, allocate arrays in device memory\n");
  
  /////////////////////////////////////
  //          Device pointers
  /////////////////////////////////////
  int *enode_d, *color_reord_d, *colornum_d, *color_d;
  float *node_val_d,*node_old_d,*edge_val_d;
  int *block_reord_d;
  
  checkCudaErrors( hipMalloc((void**)&enode_d, 2*nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&color_reord_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&color_d, nedge*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&colornum_d, c.numblock*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&edge_val_d, nedge*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&node_old_d, nnode*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&node_val_d, nnode*sizeof(float)) );
  checkCudaErrors( hipMalloc((void**)&block_reord_d, c.numblock*sizeof(int)) );
  
  checkCudaErrors( hipMemcpy(enode_d, enode, 2*nedge*sizeof(int),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(color_reord_d, c.color_reord,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(color_d, c.reordcolor,
                               nedge*sizeof(int), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(colornum_d, c.colornum, c.numblock*sizeof(int),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(edge_val_d, edge_val, nedge*sizeof(float),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(node_val_d, node_val, nnode*sizeof(float),
                              hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(block_reord_d, bc.color_reord,
                               c.numblock*sizeof(int),
                               hipMemcpyHostToDevice) );

  ///////////////////////////////////////////////////////////
  //                      Start
  ///////////////////////////////////////////////////////////
  printf("start edge based on CPU niter: %d, nnode:%d, nedge:%d, numblock: %d\n",niter,
     nnode,nedge, c.numblock);
  //   timer
  TIMER_START(total)
  //______________________________main_loop_____________________________
  for(int i=0;i<=niter;++i){
    //save old
    TIMER_START(ssol)
    ssoln<<<(nnode-1)/BLOCKSIZE+1,BLOCKSIZE>>>(node_old_d,node_val_d, nnode);
    checkCudaErrors( hipDeviceSynchronize() );
    TIMER_STOP(ssol)

    //calc next step
    for(int col=0; col<bc.colornum;col++){ 
      int start = col==0?0:bc.color_offsets[col-1]; 
      int len = bc.color_offsets[col]-start;
      TIMER_START(iter)
      iter_calc<<<len,BLOCKSIZE>>>(node_old_d,
          node_val_d, edge_val_d, enode_d, color_reord_d, nedge, color_d,
          colornum_d, block_reord_d, start);
      checkCudaErrors( hipDeviceSynchronize() );
      TIMER_STOP(iter)
    }

    // rms
    if(i%100==0){
      TIMER_START(rms)
      checkCudaErrors( hipMemcpy(node_val, node_val_d, nnode*sizeof(float),
                              hipMemcpyDeviceToHost) );
      checkCudaErrors( hipMemcpy(node_old, node_old_d, nnode*sizeof(float),
                              hipMemcpyDeviceToHost) );

      rms_calc(node_val,node_old,nnode,i);
      TIMER_STOP(rms)
    }

  }
  //____________________________end main loop___________________________
  //    timer
  TIMER_STOP(total)

  TIMER_PRINT(ssol)
  TIMER_PRINT(iter)
  TIMER_PRINT(rms)
  TIMER_PRINT(total)
  TIMER_PRINT(color)
  TIMER_PRINT(color2)

  //free
  free(enode);
  free(node_old);
  free(node_val);
  free(edge_val);
  //cuda freee
  checkCudaErrors( hipFree(enode_d) );
  checkCudaErrors( hipFree(color_reord_d) );
  checkCudaErrors( hipFree(edge_val_d) );
  checkCudaErrors( hipFree(node_old_d) );
  checkCudaErrors( hipFree(node_val_d) );
  checkCudaErrors( hipFree(color_d) );
  checkCudaErrors( hipFree(colornum_d) );
  checkCudaErrors( hipFree(block_reord_d) );
  
  return 0;
}
