#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <functional>
#include <iostream>
#include <vector>

//#define MY_SIZE int
// using MY_SIZE = std::uint32_t;

#include "colouring.hpp"
#include "hip/hip_runtime_api.h"
#include "problem.hpp"

constexpr MY_SIZE BLOCK_SIZE = 128;

/* problem_stepGPU {{{1 */
template <unsigned Dim = 1, bool SOA = false>
__global__ void problem_stepGPU(const float *__restrict__ point_weights,
                                const float *__restrict__ edge_weights,
                                const MY_SIZE *__restrict__ edge_list,
                                const MY_SIZE *__restrict__ edge_inds,
                                float *__restrict__ out, const MY_SIZE edge_num,
                                const MY_SIZE point_num) {
  MY_SIZE id = blockIdx.x * blockDim.x + threadIdx.x;
  float inc[2*Dim];
  if (id < edge_num) {
    MY_SIZE edge_ind = edge_inds[id];
#pragma unroll
    for (MY_SIZE d = 0; d < Dim; ++d) {
      MY_SIZE ind_left, ind_right;
      if (SOA) {
        ind_left = d * point_num + edge_list[2 * edge_ind];
        ind_right = d * point_num + edge_list[2 * edge_ind + 1];
      } else {
        ind_left = edge_list[2 * edge_ind] * Dim + d;
        ind_right = edge_list[2 * edge_ind + 1] * Dim + d;
      }
      inc[d] = out[ind_right] + edge_weights[edge_ind] * point_weights[ind_left];
      inc[d+Dim] = out[ind_left] + edge_weights[edge_ind] * point_weights[ind_right];
    }
#pragma unroll
    for (MY_SIZE d = 0; d < Dim; ++d) {
      MY_SIZE ind_left,ind_right;
      if (SOA) {
        ind_left = d * point_num + edge_list[2 * edge_ind];
        ind_right = d * point_num + edge_list[2 * edge_ind + 1];
      } else {
        ind_left = edge_list[2 * edge_ind] * Dim + d;
        ind_right = edge_list[2 * edge_ind + 1] * Dim + d;
      }
      out[ind_right] = inc[d];
      out[ind_left] = inc[d+Dim];
    }
  }
}
/* 1}}} */

/* problem_stepGPUHierarchical {{{1 */
template <unsigned Dim = 1, bool SOA = false>
__global__ void problem_stepGPUHierarchical(
    const MY_SIZE *__restrict__ edge_list,
    const float *__restrict__ point_weights,
    float *__restrict__ point_weights_out,
    const float *__restrict__ edge_weights,
    const MY_SIZE *__restrict__ points_to_be_cached,
    const MY_SIZE *__restrict__ points_to_be_cached_offsets,
    const std::uint8_t *__restrict__ edge_colours,
    const std::uint8_t *__restrict__ num_edge_colours, MY_SIZE num_threads,
    const MY_SIZE num_points) {
  MY_SIZE bid = blockIdx.x;
  MY_SIZE thread_ind = bid * blockDim.x + threadIdx.x;
  MY_SIZE tid = threadIdx.x;

  MY_SIZE cache_points_offset = points_to_be_cached_offsets[bid];
  MY_SIZE num_cached_point = points_to_be_cached_offsets[bid + 1] 
    - cache_points_offset;
  
    
  extern __shared__ float shared[];
  float *point_cache = shared;
  float *point_cache_out = shared + num_cached_point;

  MY_SIZE left_ind, right_ind;

  std::uint8_t our_colour;
  if (thread_ind >= num_threads) {
    our_colour = num_edge_colours[bid];
  } else {
    our_colour = edge_colours[thread_ind];
  }

  // Cache in
  for (MY_SIZE i = 0; i < num_cached_point; i += blockDim.x) {
    for (MY_SIZE d = 0; d < Dim; ++d) {
      MY_SIZE c_ind, g_ind;
      if (i + tid < num_cached_point) {
        if (SOA) {
          g_ind = d * num_points +
                       points_to_be_cached[cache_points_offset + i + tid];
          c_ind = d * num_cached_point + (i + tid);
        } else {
          g_ind =
              points_to_be_cached[cache_points_offset + i + tid] * Dim + d;
          c_ind = (i + tid) * Dim + d;
        }
        point_cache[c_ind] = point_weights[g_ind];
        point_cache_out[c_ind] = point_weights_out[g_ind];
      }
    }
  }

  __syncthreads();

  // Computation
  float increment[Dim*2];
  if (thread_ind < num_threads) {
    for (MY_SIZE d = 0; d < Dim; ++d) {
      if (SOA) {
        left_ind = d * num_cached_point + edge_list[2 * thread_ind];
        right_ind = d * num_cached_point + edge_list[2 * thread_ind + 1];
      } else {
        left_ind = edge_list[2 * thread_ind] * Dim + d;
        right_ind = edge_list[2 * thread_ind + 1] * Dim + d;
      }
      increment[d] = point_cache[left_ind] * edge_weights[thread_ind];
      increment[d+Dim] = point_cache[right_ind] * edge_weights[thread_ind];
    }
  }

  for (MY_SIZE i = 0; i < num_edge_colours[bid]; ++i) {
    if (our_colour == i) {
      for (MY_SIZE d = 0; d < Dim; ++d) {
        point_cache_out[right_ind] += increment[d];
        point_cache_out[left_ind] += increment[d+Dim];
      }
    }
    __syncthreads();
  }

  // TODO:
  // You can use about half as much shared memory, if you do not pre-load valC,
  // but instead increment here. Perhaps an additional variant.
  // Cache out
  for (MY_SIZE i = 0; i < num_cached_point; i += blockDim.x) {
    if (i + tid < num_cached_point) {
      for (MY_SIZE d = 0; d < Dim; ++d) {
        MY_SIZE write_c_ind, write_g_ind;
        if (SOA) {
          write_g_ind =
              d * num_points +
              points_to_be_cached[cache_points_offset + i + tid];
          write_c_ind = d * num_cached_point + (i + tid);
        } else {
          write_g_ind =
              points_to_be_cached[cache_points_offset + i + tid] * Dim +
              d;
          write_c_ind = (i + tid) * Dim + d;
        }
        point_weights_out[write_g_ind] = point_cache_out[write_c_ind];
      }
    }
  }
}
/* 1}}} */

/* loopGPUEdgeCentred {{{1 */
template <unsigned Dim, bool SOA>
void Problem<Dim, SOA>::loopGPUEdgeCentred(MY_SIZE num, MY_SIZE reset_every) {
  std::vector<std::vector<MY_SIZE>> partition = graph.colourEdges();
  MY_SIZE num_of_colours = partition.size();
  MY_SIZE max_thread_num = std::max_element(partition.begin(), partition.end(),
                                            [](const std::vector<MY_SIZE> &a,
                                               const std::vector<MY_SIZE> &b) {
                                              return a.size() < b.size();
                                            })
                               ->size();
  MY_SIZE num_blocks = static_cast<MY_SIZE>(
      std::ceil(double(max_thread_num) / static_cast<double>(BLOCK_SIZE)));
  float *d_edge_weights;
  data_t<float> point_weights2(point_weights.getSize(), point_weights.getDim());
  std::copy(point_weights.begin(), point_weights.end(), point_weights2.begin());
  std::vector<MY_SIZE *> d_partition;
  for (const std::vector<MY_SIZE> &colour : partition) {
    MY_SIZE *d_colour;
    MY_SIZE mem_size = sizeof(MY_SIZE) * colour.size();
    checkCudaErrors(hipMalloc((void **)&d_colour, mem_size));
    checkCudaErrors(
        hipMemcpy(d_colour, colour.data(), mem_size, hipMemcpyHostToDevice));
    d_partition.push_back(d_colour);
  }
  point_weights.initDeviceMemory();
  point_weights2.initDeviceMemory();
  checkCudaErrors(
      hipMalloc((void **)&d_edge_weights, sizeof(float) * graph.numEdges()));
  checkCudaErrors(hipMemcpy(d_edge_weights, edge_weights,
                             sizeof(float) * graph.numEdges(),
                             hipMemcpyHostToDevice));
  graph.edge_to_node.initDeviceMemory();
  // Timer t;
  TIMER_START(t);
  for (MY_SIZE i = 0; i < num; ++i) {
    for (MY_SIZE c = 0; c < num_of_colours; ++c) {
      problem_stepGPU<Dim, SOA><<<num_blocks, BLOCK_SIZE>>>(
          point_weights.getDeviceData(), d_edge_weights,
          graph.edge_to_node.getDeviceData(), d_partition[c],
          point_weights2.getDeviceData(), partition[c].size(),
          graph.numPoints());
      checkCudaErrors(hipDeviceSynchronize());
    }
    TIMER_TOGGLE(t);
    if (reset_every && i % reset_every == reset_every - 1) {
      reset();
      // Copy to point_weights2 that is currently holding the result, the next
      // copy will put it into point_weights also.
      std::copy(point_weights.begin(), point_weights.end(),
                point_weights2.begin());
      point_weights2.flushToDevice();
    }
    TIMER_TOGGLE(t);
    checkCudaErrors(hipMemcpy(
        point_weights.getDeviceData(), point_weights2.getDeviceData(),
        sizeof(float) * graph.numPoints(), hipMemcpyDeviceToDevice));
  }
  PRINT_BANDWIDTH(t, "loopGPUEdgeCentred",
                  sizeof(float) * (2 * graph.numPoints() + graph.numEdges()) *
                      num,
                  (sizeof(float) * graph.numPoints() * 2 +  // point_weights
                   sizeof(float) * graph.numEdges() +       // d_edge_weights
                   sizeof(MY_SIZE) * graph.numEdges() * 2 + // d_edge_list
                   sizeof(MY_SIZE) * graph.numEdges() * 2   // d_partition
                   ) * num);
  std::cout << " Needed " << num_of_colours << " colours" << std::endl;
  point_weights.flushToHost();
  checkCudaErrors(hipFree(d_edge_weights));
  for (MY_SIZE i = 0; i < num_of_colours; ++i) {
    checkCudaErrors(hipFree(d_partition[i]));
  }
}
/* 1}}} */

/* loopGPUHierarchical {{{1 */
template <unsigned Dim, bool SOA>
void Problem<Dim, SOA>::loopGPUHierarchical(MY_SIZE num, MY_SIZE reset_every) {
  HierarchicalColourMemory<Dim, SOA> memory(BLOCK_SIZE, *this);
  std::vector<float *> d_edge_weights;
  std::vector<MY_SIZE *> d_read_points_to_be_cached;
  std::vector<MY_SIZE *> d_write_points_to_be_cached;
  std::vector<MY_SIZE *> d_edge_list;
  std::vector<std::uint8_t *> d_edge_colours;
  std::vector<std::uint8_t *> d_num_edge_colours;
  std::vector<MY_SIZE> shared_sizes;
  data_t<float> point_weights_out(point_weights.getSize(),
                                  point_weights.getDim());
  std::copy(point_weights.begin(), point_weights.end(),
            point_weights_out.begin());
  std::vector<MY_SIZE *> d_read_points_to_be_cached_offsets;
  std::vector<MY_SIZE *> d_write_points_to_be_cached_offsets;
  point_weights.initDeviceMemory();
  point_weights_out.initDeviceMemory();
  MY_SIZE total_cache_size = 0; // for bandwidth calculations
  float avg_num_edge_colours = 0;
  for (const typename HierarchicalColourMemory<Dim, SOA>::MemoryOfOneColour
           &memory_of_one_colour : memory.colours) {
    float *d_fptr;
    MY_SIZE *d_sptr;
    std::uint8_t *d_uptr;
    // Edge weights
    checkCudaErrors(
        hipMalloc((void **)&d_fptr,
                   sizeof(float) * memory_of_one_colour.edge_weights.size()));
    d_edge_weights.push_back(d_fptr);
    checkCudaErrors(
        hipMemcpy(d_fptr, memory_of_one_colour.edge_weights.data(),
                   sizeof(float) * memory_of_one_colour.edge_weights.size(),
                   hipMemcpyHostToDevice));
    // Read points to be cached
    checkCudaErrors(
        hipMalloc((void **)&d_sptr,
                   sizeof(MY_SIZE) *
                       memory_of_one_colour.read_points_to_be_cached.size()));
    d_read_points_to_be_cached.push_back(d_sptr);
    checkCudaErrors(hipMemcpy(
        d_sptr, memory_of_one_colour.read_points_to_be_cached.data(),
        sizeof(MY_SIZE) * memory_of_one_colour.read_points_to_be_cached.size(),
        hipMemcpyHostToDevice));
    // Read points to be cached: offsets
    checkCudaErrors(hipMalloc(
        (void **)&d_sptr,
        sizeof(MY_SIZE) *
            memory_of_one_colour.read_points_to_be_cached_offsets.size()));
    d_read_points_to_be_cached_offsets.push_back(d_sptr);
    checkCudaErrors(hipMemcpy(
        d_sptr, memory_of_one_colour.read_points_to_be_cached_offsets.data(),
        sizeof(MY_SIZE) *
            memory_of_one_colour.read_points_to_be_cached_offsets.size(),
        hipMemcpyHostToDevice));
    // Write points to be cached
    checkCudaErrors(
        hipMalloc((void **)&d_sptr,
                   sizeof(MY_SIZE) *
                       memory_of_one_colour.write_points_to_be_cached.size()));
    d_write_points_to_be_cached.push_back(d_sptr);
    checkCudaErrors(hipMemcpy(
        d_sptr, memory_of_one_colour.write_points_to_be_cached.data(),
        sizeof(MY_SIZE) * memory_of_one_colour.write_points_to_be_cached.size(),
        hipMemcpyHostToDevice));
    // Write points to be cached: offsets
    checkCudaErrors(hipMalloc(
        (void **)&d_sptr,
        sizeof(MY_SIZE) *
            memory_of_one_colour.write_points_to_be_cached_offsets.size()));
    d_write_points_to_be_cached_offsets.push_back(d_sptr);
    checkCudaErrors(hipMemcpy(
        d_sptr, memory_of_one_colour.write_points_to_be_cached_offsets.data(),
        sizeof(MY_SIZE) *
            memory_of_one_colour.write_points_to_be_cached_offsets.size(),
        hipMemcpyHostToDevice));
    // Shared memory sizes
    MY_SIZE shared_size_read = 0, shared_size_write = 0;
    for (MY_SIZE i = 1;
         i < memory_of_one_colour.read_points_to_be_cached_offsets.size();
         ++i) {
      shared_size_read = std::max<MY_SIZE>(
          shared_size_read,
          memory_of_one_colour.read_points_to_be_cached_offsets[i] -
              memory_of_one_colour.read_points_to_be_cached_offsets[i - 1]);
    }
    for (MY_SIZE i = 1;
         i < memory_of_one_colour.write_points_to_be_cached_offsets.size();
         ++i) {
      shared_size_write = std::max<MY_SIZE>(
          shared_size_write,
          memory_of_one_colour.write_points_to_be_cached_offsets[i] -
              memory_of_one_colour.write_points_to_be_cached_offsets[i - 1]);
    }
    shared_sizes.push_back(shared_size_read + shared_size_write);
    total_cache_size += memory_of_one_colour.read_points_to_be_cached.size() +
                        memory_of_one_colour.write_points_to_be_cached.size();
    // Edge list
    checkCudaErrors(
        hipMalloc((void **)&d_sptr,
                   sizeof(MY_SIZE) * memory_of_one_colour.edge_list.size()));
    d_edge_list.push_back(d_sptr);
    checkCudaErrors(
        hipMemcpy(d_sptr, memory_of_one_colour.edge_list.data(),
                   sizeof(MY_SIZE) * memory_of_one_colour.edge_list.size(),
                   hipMemcpyHostToDevice));
    // Edge colours
    checkCudaErrors(hipMalloc((void **)&d_uptr,
                               sizeof(std::uint8_t) *
                                   memory_of_one_colour.edge_colours.size()));
    d_edge_colours.push_back(d_uptr);
    checkCudaErrors(hipMemcpy(d_uptr, memory_of_one_colour.edge_colours.data(),
                               sizeof(std::uint8_t) *
                                   memory_of_one_colour.edge_colours.size(),
                               hipMemcpyHostToDevice));
    // Num edge colours
    checkCudaErrors(hipMalloc(
        (void **)&d_uptr,
        sizeof(std::uint8_t) * memory_of_one_colour.num_edge_colours.size()));
    d_num_edge_colours.push_back(d_uptr);
    checkCudaErrors(hipMemcpy(
        d_uptr, memory_of_one_colour.num_edge_colours.data(),
        sizeof(std::uint8_t) * memory_of_one_colour.num_edge_colours.size(),
        hipMemcpyHostToDevice));
    avg_num_edge_colours +=
        std::accumulate(memory_of_one_colour.num_edge_colours.begin(),
                        memory_of_one_colour.num_edge_colours.end(), 0.0f);
  }
  // -----------------------
  // -  Start computation  -
  // -----------------------
  TIMER_START(t);
  MY_SIZE total_num_blocks = 0; // for bandwidth calculations
  for (MY_SIZE iteration = 0; iteration < num; ++iteration) {
    for (MY_SIZE colour_ind = 0; colour_ind < memory.colours.size();
         ++colour_ind) {
      assert(memory.colours[colour_ind].edge_list.size() % 2 == 0);
      MY_SIZE num_threads = memory.colours[colour_ind].edge_list.size() / 2;
      MY_SIZE num_blocks = static_cast<MY_SIZE>(
          std::ceil(static_cast<double>(num_threads) / BLOCK_SIZE));
      assert(num_blocks == memory.colours[colour_ind].num_edge_colours.size());
      MY_SIZE cache_size = sizeof(float) * 2 * shared_sizes[colour_ind];
      problem_stepGPUHierarchical<Dim, SOA>
          <<<num_blocks, BLOCK_SIZE, cache_size>>>(
              d_edge_list[colour_ind], point_weights.getDeviceData(),
              point_weights_out.getDeviceData(), d_edge_weights[colour_ind],
              d_read_points_to_be_cached[colour_ind],
              d_read_points_to_be_cached_offsets[colour_ind],
              d_edge_colours[colour_ind], d_num_edge_colours[colour_ind],
              num_threads, graph.numPoints());
      checkCudaErrors(hipDeviceSynchronize());
      total_num_blocks += num_blocks;
    }
    checkCudaErrors(hipMemcpy(
        point_weights.getDeviceData(), point_weights_out.getDeviceData(),
        sizeof(float) * graph.numPoints(), hipMemcpyDeviceToDevice));
    if (reset_every && iteration % reset_every == reset_every - 1) {
      TIMER_TOGGLE(t);
      reset();
      point_weights.flushToDevice();
      std::copy(point_weights.begin(), point_weights.end(),
                point_weights_out.begin());
      point_weights_out.flushToDevice();
      TIMER_TOGGLE(t);
    }
  }
  PRINT_BANDWIDTH(
      t, "GPU HierarchicalColouring",
      num * (2 * graph.numPoints() + graph.numEdges()) * sizeof(float),
      num * (sizeof(float) * graph.numPoints() * 2 +  // point_weights
             sizeof(float) * graph.numEdges() +       // edge_weights
             sizeof(MY_SIZE) * graph.numEdges() * 2 + // edge_list
             sizeof(MY_SIZE) * total_cache_size +
             sizeof(MY_SIZE) *
                 (total_num_blocks +
                  memory.colours.size()) + // points_to_be_cached_offsets
             sizeof(std::uint8_t) * graph.numEdges() // edge_colours
             ));
  std::cout << "  reuse factor: "
            << static_cast<double>(total_cache_size) / (2 * graph.numEdges())
            << std::endl;
  avg_num_edge_colours /=
      std::ceil(static_cast<double>(graph.numEdges()) / BLOCK_SIZE);
  std::cout << "  average number of colours used: " << avg_num_edge_colours
            << std::endl;
  // ---------------
  // -  Finish up  -
  // ---------------
  point_weights.flushToHost();
  for (MY_SIZE i = 0; i < memory.colours.size(); ++i) {
    checkCudaErrors(hipFree(d_num_edge_colours[i]));
    checkCudaErrors(hipFree(d_edge_colours[i]));
    checkCudaErrors(hipFree(d_edge_list[i]));
    checkCudaErrors(hipFree(d_read_points_to_be_cached_offsets[i]));
    checkCudaErrors(hipFree(d_read_points_to_be_cached[i]));
    checkCudaErrors(hipFree(d_write_points_to_be_cached_offsets[i]));
    checkCudaErrors(hipFree(d_write_points_to_be_cached[i]));
    checkCudaErrors(hipFree(d_edge_weights[i]));
  }
}
/* 1}}} */

template <unsigned Dim = 1, bool SOA = false>
using implementation_algorithm_t = void (Problem<Dim, SOA>::*)(MY_SIZE,
                                                               MY_SIZE);

/* tests {{{1 */
void testColours() {
  Graph graph(1000, 2000);
  auto v = graph.colourEdges();

  // for (const auto &vv : v) {
  //  for (MY_SIZE a : vv) {
  //    std::cout << a << " ";
  //  }
  //  std::cout << std::endl;
  //}
  std::cout << v.size() << " " << v.at(0).size() << " " << v.at(1).size()
            << std::endl;
}

void testGPUSolution(MY_SIZE num) {
  std::cout << "CPU edge vs GPU edge" << std::endl;

  std::vector<float> result1;
  double rms = 0;
  MY_SIZE N = 1000;
  MY_SIZE M = 2000;
  MY_SIZE reset_every = 10;
  {
    srand(1);
    Problem<> problem(N, M);
    problem.loopCPUEdgeCentred(num, reset_every);
    float abs_max = 0;
    for (MY_SIZE i = 0; i < problem.graph.numPoints(); ++i) {
      result1.push_back(problem.point_weights[i]);
      abs_max = std::max(abs_max, problem.point_weights[i]);
    }
    std::cout << "Abs max: " << abs_max << std::endl;
  }

  {
    srand(1);
    Problem<> problem(N, M);
    problem.loopGPUEdgeCentred(num, reset_every);
    float abs_max = 0;
    for (MY_SIZE i = 0; i < problem.graph.numPoints(); ++i) {
      rms += std::pow(problem.point_weights[i] - result1[i], 2);
      abs_max = std::max(abs_max, problem.point_weights[i]);
    }
    rms = std::pow(rms / result1.size(), 0.5);
    std::cout << "Abs max: " << abs_max << std::endl;
    std::cout << "RMS: " << rms << std::endl;
  }
}

void testHierarchicalColouring() {
  Problem<> problem(2, 4);
  constexpr MY_SIZE BLOCK_SIZE = 3;
  constexpr bool PRINT_RESULT = true;
  if (PRINT_RESULT) {
    std::cout << "Edge weights: ";
    for (MY_SIZE i = 0; i < problem.graph.numEdges(); ++i) {
      std::cout << problem.edge_weights[i] << " ";
    }
    std::cout << std::endl << "Edge list: ";
    for (MY_SIZE i = 0; i < problem.graph.numEdges(); ++i) {
      std::cout << problem.graph.edge_to_node[2 * i] << "->"
                << problem.graph.edge_to_node[2 * i + 1] << std::endl;
    }
    std::cout << std::endl;
  }
  Timer t;
  HierarchicalColourMemory<> memory(BLOCK_SIZE, problem);
  std::cout << "memory colouring time: ";
  t.printTime();
  std::cout << std::endl;
  if (PRINT_RESULT) {
    for (const auto &c : memory.colours) {
      std::cout << "================================================"
                << std::endl
                << "Memory:" << std::endl;
      std::cout << "Edge weights: ";
      for (float w : c.edge_weights) {
        std::cout << w << " ";
      }
      std::cout << std::endl << "Read points to be cached: ";
      for (const auto &p : c.read_points_to_be_cached) {
        std::cout << p << " ";
      }
      std::cout << std::endl << "Read points to be cached (offsets): ";
      for (const auto &p : c.read_points_to_be_cached_offsets) {
        std::cout << p << " ";
      }
      std::cout << std::endl << "Write points to be cached: ";
      for (const auto &p : c.write_points_to_be_cached) {
        std::cout << p << " ";
      }
      std::cout << std::endl << "Write points to be cached (offsets): ";
      for (const auto &p : c.write_points_to_be_cached_offsets) {
        std::cout << p << " ";
      }
      std::cout << std::endl << "Edge list: ";
      for (MY_SIZE i = 0; i < c.edge_list.size(); i += 2) {
        std::cout << c.edge_list[i] << "->" << c.edge_list[i + 1] << " ";
      }
      std::cout << std::endl;
      std::cout << "Num of edge colours: ";
      for (std::uint8_t nec : c.num_edge_colours) {
        std::cout << static_cast<unsigned>(nec) << " ";
      }
      std::cout << std::endl;
      std::cout << "Edge colours: ";
      for (std::uint8_t cc : c.edge_colours) {
        std::cout << static_cast<int>(cc) << " ";
      }
      std::cout << std::endl;
    }
  }
}

void testGPUHierarchicalSolution(MY_SIZE num) {
  std::cout << "CPU edge vs GPU hierarchical" << std::endl;

  std::vector<float> result1;
  double rms = 0;
  MY_SIZE N = 1000;
  MY_SIZE M = 2000;
  MY_SIZE reset_every = 10;
  {
    srand(1);
    Problem<> problem(N, M);
    problem.loopCPUEdgeCentred(num, reset_every);
    float abs_max = 0;
    for (MY_SIZE i = 0; i < problem.graph.numPoints(); ++i) {
      result1.push_back(problem.point_weights[i]);
      abs_max = std::max(abs_max, problem.point_weights[i]);
    }
    std::cout << "Abs max: " << abs_max << std::endl;
  }

  {
    srand(1);
    Problem<> problem(N, M);
    problem.loopGPUHierarchical(num, reset_every);
    float abs_max = 0;
    for (MY_SIZE i = 0; i < problem.graph.numPoints(); ++i) {
      rms += std::pow(problem.point_weights[i] - result1[i], 2);
      abs_max = std::max(abs_max, problem.point_weights[i]);
    }
    std::cout << "Abs max: " << abs_max << std::endl;
    rms = std::pow(rms / result1.size(), 0.5);
    std::cout << "RMS: " << rms << std::endl;
  }
}

void testTwoImplementations(MY_SIZE num, MY_SIZE N, MY_SIZE M,
                            MY_SIZE reset_every,
                            implementation_algorithm_t<> algorithm1,
                            implementation_algorithm_t<> algorithm2) {
  std::vector<float> result1;
  double rms = 0;
  {
    srand(1);
    Problem<> problem(N, M);
    (problem.*algorithm1)(num, reset_every);
    float abs_max = 0;
    for (MY_SIZE i = 0; i < problem.graph.numPoints(); ++i) {
      result1.push_back(problem.point_weights[i]);
      abs_max = std::max(abs_max, problem.point_weights[i]);
    }
    std::cout << "Abs max: " << abs_max << std::endl;
  }

  {
    srand(1);
    Problem<> problem(N, M);
    (problem.*algorithm2)(num, reset_every);
    float abs_max = 0;
    for (MY_SIZE i = 0; i < problem.graph.numPoints(); ++i) {
      rms += std::pow(problem.point_weights[i] - result1[i], 2);
      abs_max = std::max(abs_max, problem.point_weights[i]);
    }
    std::cout << "Abs max: " << abs_max << std::endl;
    rms = std::pow(rms / result1.size(), 0.5);
    std::cout << "RMS: " << rms << std::endl;
  }
}

void testReordering(MY_SIZE num, MY_SIZE N, MY_SIZE M, MY_SIZE reset_every,
                    implementation_algorithm_t<> algorithm1,
                    implementation_algorithm_t<> algorithm2) {
  std::vector<float> result1;
  double rms = 0;
  {
    srand(1);
    Problem<> problem(N, M);
    /*std::ifstream f("test.in");*/
    /*Problem<> problem (f);*/
    std::cout << "Problem 1 created" << std::endl;
    problem.reorder();
    std::cout << "Problem 1 reordered" << std::endl;
    (problem.*algorithm1)(num, reset_every);
    float abs_max = 0;
    for (MY_SIZE i = 0; i < problem.graph.numPoints(); ++i) {
      result1.push_back(problem.point_weights[i]);
      abs_max = std::max(abs_max, std::abs(problem.point_weights[i]));
    }
    std::cout << "Abs max: " << abs_max << std::endl;
  }

  {
    srand(1);
    Problem<> problem(N, M);
    /*std::ifstream f("rotor37_mesh");*/
    /*Problem<> problem (f);*/
    std::cout << "Problem 2 created" << std::endl;
    (problem.*algorithm2)(num, reset_every);
    problem.reorder();
    float abs_max = 0;
    for (MY_SIZE i = 0; i < problem.graph.numPoints(); ++i) {
      rms += std::pow(problem.point_weights[i] - result1[i], 2);
      abs_max = std::max(abs_max, std::abs(problem.point_weights[i]));
    }
    std::cout << "Abs max: " << abs_max << std::endl;
    rms = std::pow(rms / result1.size(), 0.5);
    std::cout << "RMS: " << rms << std::endl;
  }
}
/* 1}}} */

void generateTimes(std::string in_file) {
  constexpr MY_SIZE num = 500;
  std::cout << ":::: Generating problems from file: " << in_file
            << "::::" << std::endl;
  std::function<void(implementation_algorithm_t<>)> run =
      [&in_file](implementation_algorithm_t<> algo) {
        std::ifstream f(in_file);
        Problem<> problem(f);
        std::cout << "--Problem created" << std::endl;
        (problem.*algo)(num, 0);
        std::cout << "--Problem finished." << std::endl;
      };
  run(&Problem<>::loopCPUEdgeCentred);
  run(&Problem<>::loopCPUEdgeCentredOMP);
  run(&Problem<>::loopGPUEdgeCentred);
  run(&Problem<>::loopGPUHierarchical);
  std::cout << "Finished." << std::endl;
}

int main(int argc, const char **argv) {
  findCudaDevice(argc, argv);
  /*generateTimes("grid_513x513_default");*/
  /*generateTimes("grid_513x513_rcm");*/
  /*generateTimes("grid_513x513_scotch");*/
  /*generateTimes("grid_513x513_hardcoded2");*/
  /*generateTimes("rotor37_nonrenum");*/
  /*generateTimes("rotor37_nonrenum.rcm");*/
  /*generateTimes("rotor37_nonrenum.scotch");*/
  /*generateTimes("grid_1025x1025_default");*/
  /*generateTimes("grid_1025x1025_default.rcm");*/
  /*generateTimes("grid_1025x1025_default.scotch");*/
  /*generateTimes("grid_1025x1025_hardcoded2");*/
  MY_SIZE num = 1000;
  MY_SIZE N = 1000, M = 200;
  MY_SIZE reset_every = 1001;
  testTwoImplementations(num, N, M, reset_every,
                         &Problem<>::loopCPUEdgeCentredOMP,
                         &Problem<>::loopGPUHierarchical);
  hipDeviceReset();
}

// vim:set et sw=2 ts=2 fdm=marker:
