#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <functional>
#include <iostream>
#include <set>
#include <vector>

#include "colouring.hpp"
#include "hip/hip_runtime_api.h"
#include "kernels/mine.hpp"
#include "problem.hpp"
#include "tests.hpp"

/* copyKernels {{{1 */
__global__ void copyKernel(const float *__restrict__ a, float *__restrict__ b,
                           MY_SIZE size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const float4 *__restrict__ a_ = reinterpret_cast<const float4 *>(a);
  float4 *__restrict__ b_ = reinterpret_cast<float4 *>(b);
  if (tid * 4 < size) {
    b_[tid] = a_[tid];
  }
}

__global__ void copyKernel(const double *__restrict__ a, double *__restrict__ b,
                           MY_SIZE size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const double2 *__restrict__ a_ = reinterpret_cast<const double2 *>(a);
  double2 *__restrict__ b_ = reinterpret_cast<double2 *>(b);
  if (tid * 2 < size) {
    b_[tid] = a_[tid];
  }
}
/* 1}}} */

template <unsigned Dim = 1, bool SOA = false, typename DataType = float,
          class ForwardIterator>
size_t countCacheLinesForBlock(ForwardIterator block_begin,
                               ForwardIterator block_end) {
  std::set<MY_SIZE> cache_lines;
  MY_SIZE data_per_cacheline = 32 / sizeof(DataType);

  for (; block_begin != block_end; ++block_begin) {
    MY_SIZE point_id = *block_begin;
    MY_SIZE cache_line_id = SOA ? point_id / data_per_cacheline
                                : point_id * Dim / data_per_cacheline;
    if (!SOA) {
      if (data_per_cacheline / Dim > 0) {
        assert(data_per_cacheline % Dim == 0);
        cache_lines.insert(cache_line_id);
      } else {
        assert(Dim % data_per_cacheline == 0);
        MY_SIZE cache_line_per_data =
            Dim / data_per_cacheline; // Assume that Dim is multiple of
                                      // data_per_cacheline
        for (MY_SIZE i = 0; i < cache_line_per_data; ++i) {
          cache_lines.insert(cache_line_id++);
        }
      }
    } else {
      cache_lines.insert(cache_line_id);
    }
  }
  return (SOA ? Dim : 1) * cache_lines.size();
}

/* loopGPUCellCentred {{{1 */
template <unsigned PointDim, unsigned CellDim, bool SOA, typename DataType>
template <class UserFunc>
void Problem<PointDim, CellDim, SOA, DataType>::loopGPUCellCentred(
    MY_SIZE num) {
  std::vector<std::vector<MY_SIZE>> partition = mesh.colourCells();
  MY_SIZE num_of_colours = partition.size();
  assert(num_of_colours > 0);
  data_t point_weights2(
      data_t::create<DataType>(point_weights.getSize(), PointDim));
  std::copy(point_weights.begin(), point_weights.end(), point_weights2.begin());
  std::vector<data_t> d_cell_lists;
  std::vector<data_t> d_cell_weights;
  MY_SIZE total_num_cache_lines = 0;
  MY_SIZE total_num_blocks = 0;
  for (const std::vector<MY_SIZE> &colour : partition) {
    d_cell_lists.emplace_back(data_t::create<MY_SIZE>(colour.size(), MESH_DIM));
    d_cell_weights.emplace_back(
        data_t::create<DataType>(colour.size(), CellDim));
    for (std::size_t i = 0; i < colour.size(); ++i) {
      std::copy_n(mesh.cell_to_node.begin<MY_SIZE>() + MESH_DIM * colour[i],
                  MESH_DIM,
                  d_cell_lists.back().begin<MY_SIZE>() + MESH_DIM * i);
      for (unsigned d = 0; d < CellDim; ++d) {
        d_cell_weights.back().operator[]<DataType>(
            index<true>(colour.size(), i, CellDim, d)) =
            cell_weights.operator[]<DataType>(
                index<true>(mesh.numCells(), colour[i], CellDim, d));
      }
    }
    d_cell_lists.back().initDeviceMemory();
    d_cell_weights.back().initDeviceMemory();
    MY_SIZE num_blocks = std::ceil(static_cast<double>(colour.size()) /
                                   static_cast<double>(block_size));
    total_num_blocks += num_blocks;
    for (MY_SIZE i = 0; i < num_blocks; ++i) {
      total_num_cache_lines += countCacheLinesForBlock<PointDim, SOA, DataType>(
          d_cell_lists.back().begin<MY_SIZE>() + MESH_DIM * block_size * i,
          d_cell_lists.back().begin<MY_SIZE>() +
              MESH_DIM *
                  std::min<MY_SIZE>(colour.size(), block_size * (i + 1)));
    }
  }
  point_weights.initDeviceMemory();
  point_weights2.initDeviceMemory();
  CUDA_TIMER_START(t);
  for (MY_SIZE i = 0; i < num; ++i) {
    for (MY_SIZE c = 0; c < num_of_colours; ++c) {
      MY_SIZE num_blocks = std::ceil(static_cast<double>(partition[c].size()) /
                                     static_cast<double>(block_size));
      /*problem_stepGPU<PointDim, CellDim, SOA, DataType,*/
      /*                MESH_DIM><<<num_blocks, block_size>>>(*/
      /*    point_weights.getDeviceData<DataType>(),*/
      /*    d_cell_weights[c].getDeviceData<DataType>(),*/
      /*    d_cell_lists[c].getDeviceData<MY_SIZE>(),*/
      /*    point_weights2.getDeviceData<DataType>(), partition[c].size(),*/
      /*    mesh.numPoints(), mesh.numCells());*/
      UserFunc::template call<SOA>(
          point_weights.getDeviceData(), point_weights2.getDeviceData(),
          d_cell_weights[c].getDeviceData(),
          d_cell_lists[c].getDeviceData<MY_SIZE>(), partition[c].size(),
          mesh.numPoints(), partition[c].size(), num_blocks, block_size);
      checkCudaErrors(hipDeviceSynchronize());
    }
    TIMER_TOGGLE(t);
    checkCudaErrors(hipMemcpy(point_weights.getDeviceData<DataType>(),
                               point_weights2.getDeviceData<DataType>(),
                               sizeof(DataType) * mesh.numPoints() * PointDim,
                               hipMemcpyDeviceToDevice));
    TIMER_TOGGLE(t);
  }
  PRINT_BANDWIDTH(
      t, "loopGPUCellCentred",
      (sizeof(DataType) *
           (2.0 * PointDim * mesh.numPoints() + CellDim * mesh.numCells()) +
       1.0 * MESH_DIM * sizeof(MY_SIZE) * mesh.numCells()) *
          num,
      (sizeof(DataType) * mesh.numPoints() * PointDim * 2.0 + // point_weights
       sizeof(DataType) * mesh.numCells() * CellDim * 1.0 +   // d_cell_weights
       1.0 * sizeof(MY_SIZE) * mesh.numCells() * MESH_DIM     // d_cell_list
       ) * num);
  std::cout << " Needed " << num_of_colours << " colours" << std::endl;
  std::cout << "  average cache_line / block: "
            << static_cast<double>(total_num_cache_lines) / total_num_blocks
            << std::endl;
  PRINT_BANDWIDTH(
      t, " -cache line",
      num * (total_num_cache_lines * 32.0 * 2 +
             1.0 * CellDim * mesh.numCells() * sizeof(DataType) +
             1.0 * MESH_DIM * mesh.numCells() * sizeof(MY_SIZE)),
      num *
          (2 * 32.0 * total_num_cache_lines + // indirect accessed cache lines
           sizeof(DataType) * mesh.numCells() * CellDim * 1.0 + // cell_weights
           1.0 * sizeof(MY_SIZE) * mesh.numCells() * MESH_DIM   // cell_list
           ));
  point_weights.flushToHost();
}
/* 1}}} */

/* loopGPUHierarchical {{{1 */
template <unsigned PointDim, unsigned CellDim, bool SOA, typename DataType>
template <class UserFunc>
void Problem<PointDim, CellDim, SOA, DataType>::loopGPUHierarchical(
    MY_SIZE num) {
  TIMER_START(t_colouring);
  HierarchicalColourMemory<MESH_DIM, PointDim, CellDim, SOA, DataType> memory(
      *this, partition_vector);
  TIMER_PRINT(t_colouring, "Hierarchical colouring: colouring");
  const auto d_memory = memory.getDeviceMemoryOfOneColour();
  data_t point_weights_out(
      data_t::create<DataType>(point_weights.getSize(), PointDim));
  std::copy(point_weights.begin(), point_weights.end(),
            point_weights_out.begin());
  point_weights.initDeviceMemory();
  point_weights_out.initDeviceMemory();
  MY_SIZE total_cache_size = 0; // for bandwidth calculations
  DataType avg_num_cell_colours = 0;
  MY_SIZE total_num_blocks = 0;
  MY_SIZE total_shared_size = 0;
  size_t total_num_cache_lines = 0;
  for (MY_SIZE i = 0; i < memory.colours.size(); ++i) {
    const typename HierarchicalColourMemory<MESH_DIM, PointDim, CellDim, SOA,
                                            DataType>::MemoryOfOneColour
        &memory_of_one_colour = memory.colours[i];
    MY_SIZE num_threads = memory_of_one_colour.cell_list.size() / MESH_DIM;
    MY_SIZE num_blocks = static_cast<MY_SIZE>(
        std::ceil(static_cast<double>(num_threads) / block_size));
    total_cache_size += memory_of_one_colour.points_to_be_cached.size();
    avg_num_cell_colours +=
        std::accumulate(memory_of_one_colour.num_cell_colours.begin(),
                        memory_of_one_colour.num_cell_colours.end(), 0.0f);
    total_num_blocks += num_blocks;
    total_shared_size += num_blocks * d_memory[i].shared_size;
    for (MY_SIZE j = 0;
         j < memory_of_one_colour.points_to_be_cached_offsets.size() - 1; ++j) {
      total_num_cache_lines +=
          countCacheLinesForBlock<PointDim, SOA, DataType,
                                  std::vector<MY_SIZE>::const_iterator>(
              memory_of_one_colour.points_to_be_cached.begin() +
                  memory_of_one_colour.points_to_be_cached_offsets[j],
              memory_of_one_colour.points_to_be_cached.begin() +
                  memory_of_one_colour.points_to_be_cached_offsets[j + 1]);
    }
  }
  // -----------------------
  // -  Start computation  -
  // -----------------------
  CUDA_TIMER_START(timer_calc);
  TIMER_TOGGLE(timer_calc);
  CUDA_TIMER_START(timer_copy);
  TIMER_TOGGLE(timer_copy);
  for (MY_SIZE iteration = 0; iteration < num; ++iteration) {
    for (MY_SIZE colour_ind = 0; colour_ind < memory.colours.size();
         ++colour_ind) {
      assert(memory.colours[colour_ind].cell_list.size() % MESH_DIM == 0);
      MY_SIZE num_threads =
          memory.colours[colour_ind].cell_list.size() / MESH_DIM;
      MY_SIZE num_blocks = memory.colours[colour_ind].num_cell_colours.size();
      assert(num_blocks == memory.colours[colour_ind].block_offsets.size() - 1);
      // + 32 in case it needs to avoid shared mem bank collisions
      MY_SIZE cache_size =
          sizeof(DataType) * (d_memory[colour_ind].shared_size + 32) * PointDim;
      TIMER_TOGGLE(timer_calc);
      UserFunc::template call<SOA>(
          point_weights.getDeviceData(), point_weights_out.getDeviceData(),
          static_cast<MY_SIZE *>(d_memory[colour_ind].points_to_be_cached),
          static_cast<MY_SIZE *>(
              d_memory[colour_ind].points_to_be_cached_offsets),
          d_memory[colour_ind].cell_weights,
          static_cast<MY_SIZE *>(d_memory[colour_ind].cell_list),
          static_cast<std::uint8_t *>(d_memory[colour_ind].num_cell_colours),
          static_cast<std::uint8_t *>(d_memory[colour_ind].cell_colours),
          static_cast<MY_SIZE *>(d_memory[colour_ind].block_offsets),
          num_threads, mesh.numPoints(), num_threads, num_blocks, block_size,
          cache_size);
      TIMER_TOGGLE(timer_calc);
      checkCudaErrors(hipDeviceSynchronize());
    }
    MY_SIZE copy_size = mesh.numPoints() * PointDim;
    TIMER_TOGGLE(timer_copy);
    MY_SIZE num_copy_blocks = std::ceil(static_cast<float>(copy_size) / 512.0);
    copyKernel<<<num_copy_blocks, 512>>>(
        point_weights_out.getDeviceData<DataType>(),
        point_weights.getDeviceData<DataType>(), copy_size);
    TIMER_TOGGLE(timer_copy);
  }
  PRINT_BANDWIDTH(
      timer_calc, "GPU HierarchicalColouring",
      num * ((2.0 * PointDim * mesh.numPoints() + CellDim * mesh.numCells()) *
                 sizeof(DataType) +
             1.0 * MESH_DIM * mesh.numCells() * sizeof(MY_SIZE)),
      num *
          (sizeof(DataType) * mesh.numPoints() * PointDim *
               2.0 +                                            // point_weights
           sizeof(DataType) * mesh.numCells() * CellDim * 1.0 + // cell_weights
           sizeof(MY_SIZE) * mesh.numCells() * 1.0 * MESH_DIM + // cell_list
           sizeof(MY_SIZE) * total_cache_size * 1.0 +
           sizeof(MY_SIZE) *
               (total_num_blocks * 1.0 +
                memory.colours.size()) + // points_to_be_cached_offsets
           sizeof(MY_SIZE) * (total_num_blocks * 1.0) + // block_offsets
           sizeof(std::uint8_t) * mesh.numCells()       // cell_colours
           ));
  PRINT_BANDWIDTH(timer_copy, " -copy",
                  2.0 * num * sizeof(DataType) * PointDim * mesh.numPoints(),
                  2.0 * num * sizeof(DataType) * PointDim * mesh.numPoints());
  std::cout << "  reuse factor: "
            << static_cast<double>(total_cache_size) /
                   (MESH_DIM * mesh.numCells())
            << std::endl;
  std::cout
      << "  cache/shared mem: "
      << static_cast<double>(total_cache_size) / total_shared_size
      << "\n  shared mem reuse factor (total shared / (MeshDim * #cells)): "
      << static_cast<double>(total_shared_size) / (MESH_DIM * mesh.numCells())
      << std::endl;
  std::cout << "  average cache_line / block: "
            << static_cast<double>(total_num_cache_lines) / total_num_blocks
            << std::endl;
  PRINT_BANDWIDTH(
      timer_calc, " -cache line",
      num * (total_num_cache_lines * 32.0 * 2 +
             1.0 * CellDim * mesh.numCells() * sizeof(DataType) +
             1.0 * MESH_DIM * mesh.numCells() * sizeof(MY_SIZE)),
      num *
          (2 * 32.0 * total_num_cache_lines + // indirect accessed cache lines
           sizeof(DataType) * mesh.numCells() * CellDim * 1.0 + // cell_weights
           sizeof(MY_SIZE) * mesh.numCells() * 1.0 * MESH_DIM + // cell_list
           sizeof(MY_SIZE) * total_cache_size * 1.0 +
           sizeof(MY_SIZE) *
               (total_num_blocks * 1.0 +
                memory.colours.size()) + // points_to_be_cached_offsets
           sizeof(MY_SIZE) * (total_num_blocks * 1.0) + // block_offsets
           sizeof(std::uint8_t) * mesh.numCells()       // cell_colours
           ));
  avg_num_cell_colours /= total_num_blocks;
  std::cout << "  average number of colours used: " << avg_num_cell_colours
            << std::endl;
  // ---------------
  // -  Finish up  -
  // ---------------
  point_weights.flushToHost();
}
/* 1}}} */

template <unsigned PointDim = 1, unsigned CellDim = 1, bool SOA = false,
          bool RunCPU = true, typename DataType = float>
void generateTimes(std::string in_file) {
  constexpr MY_SIZE num = 500;
  std::cout << ":::: Generating problems from file: " << in_file
            << "::::" << std::endl
            << "     Point dimension: " << PointDim
            << " Cell dimension: " << CellDim << " SOA: " << std::boolalpha
            << SOA << "\n     Data type: "
            << (sizeof(DataType) == sizeof(float) ? "float" : "double")
            << std::endl;
  std::function<void(
      implementation_algorithm_t<PointDim, CellDim, SOA, DataType>, MY_SIZE)>
      run = [&in_file](
          implementation_algorithm_t<PointDim, CellDim, SOA, DataType> algo,
          MY_SIZE num) {
        std::ifstream f(in_file);
        Problem<PointDim, CellDim, SOA, DataType> problem(f, 288);
        if (in_file.find("metis") < in_file.size()) {
          std::ifstream f_part(in_file + "_part");
          problem.readPartition(f_part);
          problem.reorderToPartition();
          problem.renumberPoints();
        }
        std::cout << "--Problem created" << std::endl;
        (problem.*algo)(num);
        std::cout << "--Problem finished." << std::endl;
      };
  run(&Problem<PointDim, CellDim, SOA,
               DataType>::template loopCPUCellCentred<MINE_KERNEL(StepSeq)>,
      RunCPU ? num : 1);
  run(&Problem<PointDim, CellDim, SOA,
               DataType>::template loopCPUCellCentredOMP<MINE_KERNEL(StepOMP)>,
      RunCPU ? num : 1);
  run(&Problem<PointDim, CellDim, SOA, DataType>::loopGPUCellCentred, num);
  run(&Problem<PointDim, CellDim, SOA, DataType>::loopGPUHierarchical, num);
  std::cout << "Finished." << std::endl;
}

template <unsigned PointDim = 1, unsigned CellDim = 1, bool SOA = false,
          typename DataType = float>
void generateTimesWithBlockDims(MY_SIZE N, MY_SIZE M,
                                std::pair<MY_SIZE, MY_SIZE> block_dims) {
  constexpr MY_SIZE num = 500;
  MY_SIZE block_size = block_dims.first == 0
                           ? block_dims.second
                           : block_dims.first * block_dims.second * 2;
  std::cout << ":::: Generating problems with block size: " << block_dims.first
            << "x" << block_dims.second << " (= " << block_size << ")"
            << "::::" << std::endl
            << "     Point dimension: " << PointDim
            << " Cell dimension: " << CellDim << " SOA: " << std::boolalpha
            << SOA << "\n     Data type: "
            << (sizeof(DataType) == sizeof(float) ? "float" : "double")
            << std::endl;
  std::function<void(
      implementation_algorithm_t<PointDim, CellDim, SOA, DataType>)>
      run = [&](
          implementation_algorithm_t<PointDim, CellDim, SOA, DataType> algo) {
        Problem<PointDim, CellDim, SOA, DataType> problem(
            std::move(StructuredProblem<PointDim, CellDim, SOA, DataType>(
                N, M, block_dims)));
        std::cout << "--Problem created" << std::endl;
        (problem.*algo)(num);
        std::cout << "--Problem finished." << std::endl;
      };
  run(&Problem<PointDim, CellDim, SOA, DataType>::loopGPUCellCentred);
  run(&Problem<PointDim, CellDim, SOA, DataType>::loopGPUHierarchical);
  std::cout << "Finished." << std::endl;
}

template <unsigned PointDim = 1, unsigned CellDim = 1, bool SOA = false,
          typename DataType = float>
void generateTimesDifferentBlockDims(MY_SIZE N, MY_SIZE M) {
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {0, 32});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {2, 8});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {4, 4});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {0, 128});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {2, 32});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {4, 16});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {8, 8});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {0, 288});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {2, 72});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {4, 36});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {12, 12});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {9, 8});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {0, 512});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {2, 128});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {4, 64});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {8, 32});
  generateTimesWithBlockDims<PointDim, CellDim, SOA, DataType>(N, M, {16, 16});
}

void generateTimesFromFile(int argc, const char **argv) {
  if (argc <= 1) {
    std::cerr << "Usage: " << argv[0] << " <input mesh>" << std::endl;
    std::exit(1);
  }
  // AOS
  generateTimes<1, 1, false, false>(argv[1]);
  generateTimes<4, 1, false, false>(argv[1]);
  generateTimes<8, 1, false, false>(argv[1]);
  generateTimes<16, 1, false, false>(argv[1]);
  generateTimes<1, 1, false, false>(argv[1]);
  generateTimes<4, 4, false, false>(argv[1]);
  generateTimes<8, 8, false, false>(argv[1]);
  generateTimes<16, 16, false, false>(argv[1]);
  generateTimes<1, 1, false, false, double>(argv[1]);
  generateTimes<4, 1, false, false, double>(argv[1]);
  generateTimes<8, 1, false, false, double>(argv[1]);
  generateTimes<16, 1, false, false, double>(argv[1]);
  generateTimes<1, 1, false, false, double>(argv[1]);
  generateTimes<4, 4, false, false, double>(argv[1]);
  generateTimes<8, 8, false, false, double>(argv[1]);
  generateTimes<16, 16, false, false, double>(argv[1]);
  // SOA
  generateTimes<1, 1, true, false>(argv[1]);
  generateTimes<4, 1, true, false>(argv[1]);
  generateTimes<8, 1, true, false>(argv[1]);
  generateTimes<16, 1, true, false>(argv[1]);
  generateTimes<1, 1, true, false>(argv[1]);
  generateTimes<4, 4, true, false>(argv[1]);
  generateTimes<8, 8, true, false>(argv[1]);
  generateTimes<16, 16, true, false>(argv[1]);
  generateTimes<1, 1, true, false, double>(argv[1]);
  generateTimes<4, 1, true, false, double>(argv[1]);
  generateTimes<8, 1, true, false, double>(argv[1]);
  generateTimes<16, 1, true, false, double>(argv[1]);
  generateTimes<1, 1, true, false, double>(argv[1]);
  generateTimes<4, 4, true, false, double>(argv[1]);
  generateTimes<8, 8, true, false, double>(argv[1]);
  generateTimes<16, 16, true, false, double>(argv[1]);
}

void testReordering() {
  MY_SIZE num = 500;
  MY_SIZE N = 100, M = 200;
  constexpr unsigned TEST_DIM = 4;
  constexpr unsigned TEST_CELL_DIM = 4;
  testReordering<TEST_DIM, TEST_CELL_DIM, false, float>(
      num, N, M,
      &Problem<TEST_DIM, TEST_CELL_DIM, false, float>::loopCPUCellCentredOMP<
          MINE_KERNEL(StepOMP) < TEST_DIM, TEST_CELL_DIM, float>>,
      &Problem<TEST_DIM, TEST_CELL_DIM, false, float>::loopCPUCellCentredOMP<
          MINE_KERNEL(StepOMP) < TEST_DIM, TEST_CELL_DIM, float>>);
  testReordering<TEST_DIM, TEST_CELL_DIM, true, float>(
      num, N, M,
      &Problem<TEST_DIM, TEST_CELL_DIM, true, float>::loopCPUCellCentredOMP<
          MINE_KERNEL(StepOMP) < TEST_DIM, TEST_CELL_DIM, float>>,
      &Problem<TEST_DIM, TEST_CELL_DIM, true, float>::loopCPUCellCentredOMP<
          MINE_KERNEL(StepOMP) < TEST_DIM, TEST_CELL_DIM, float>>);
}

/*void testPartitioning() {*/
/*  MY_SIZE num = 500;*/
/*  MY_SIZE N = 100, M = 200;*/
/*  constexpr unsigned TEST_DIM = 4;*/
/*  constexpr unsigned TEST_CELL_DIM = 4;*/
/*  testPartitioning<TEST_DIM, TEST_CELL_DIM, false, float>(num, N, M);*/
/*  testPartitioning<TEST_DIM, TEST_CELL_DIM, true, float>(num, N, M);*/
/*}*/

void generateTimesDifferentBlockDims() {
  // SOA
  generateTimesDifferentBlockDims<1, 1, true, float>(1153, 1153);
  generateTimesDifferentBlockDims<2, 1, true, float>(1153, 1153);
  generateTimesDifferentBlockDims<4, 1, true, float>(1153, 1153);
  generateTimesDifferentBlockDims<8, 1, true, float>(1153, 1153);
  generateTimesDifferentBlockDims<1, 1, true, float>(1153, 1153);
  generateTimesDifferentBlockDims<2, 2, true, float>(1153, 1153);
  generateTimesDifferentBlockDims<4, 4, true, float>(1153, 1153);
  generateTimesDifferentBlockDims<8, 8, true, float>(1153, 1153);
  // AOS
  generateTimesDifferentBlockDims<1, 1, false, float>(1153, 1153);
  generateTimesDifferentBlockDims<2, 1, false, float>(1153, 1153);
  generateTimesDifferentBlockDims<4, 1, false, float>(1153, 1153);
  generateTimesDifferentBlockDims<8, 1, false, float>(1153, 1153);
  generateTimesDifferentBlockDims<1, 1, false, float>(1153, 1153);
  generateTimesDifferentBlockDims<2, 2, false, float>(1153, 1153);
  generateTimesDifferentBlockDims<4, 4, false, float>(1153, 1153);
  generateTimesDifferentBlockDims<8, 8, false, float>(1153, 1153);
  // SOA
  generateTimesDifferentBlockDims<1, 1, true, double>(1153, 1153);
  generateTimesDifferentBlockDims<2, 1, true, double>(1153, 1153);
  generateTimesDifferentBlockDims<4, 1, true, double>(1153, 1153);
  generateTimesDifferentBlockDims<8, 1, true, double>(1153, 1153);
  generateTimesDifferentBlockDims<1, 1, true, double>(1153, 1153);
  generateTimesDifferentBlockDims<2, 2, true, double>(1153, 1153);
  generateTimesDifferentBlockDims<4, 4, true, double>(1153, 1153);
  generateTimesDifferentBlockDims<8, 8, true, double>(1153, 1153);
  // AOS
  generateTimesDifferentBlockDims<1, 1, false, double>(1153, 1153);
  generateTimesDifferentBlockDims<2, 1, false, double>(1153, 1153);
  generateTimesDifferentBlockDims<4, 1, false, double>(1153, 1153);
  generateTimesDifferentBlockDims<8, 1, false, double>(1153, 1153);
  generateTimesDifferentBlockDims<1, 1, false, double>(1153, 1153);
  generateTimesDifferentBlockDims<2, 2, false, double>(1153, 1153);
  generateTimesDifferentBlockDims<4, 4, false, double>(1153, 1153);
  generateTimesDifferentBlockDims<8, 8, false, double>(1153, 1153);
}

int main(int argc, const char **argv) {
  /*generateTimesFromFile(argc, argv);*/
  testImplementations();
  /*testReordering();*/
  /*testPartitioning();*/
  /*generateTimesDifferentBlockDims();*/
  /*measurePartitioning();*/
  return 0;
}

// vim:set et sw=2 ts=2 fdm=marker:
