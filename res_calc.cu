#include "hip/hip_runtime.h"
#include <algorithm>
#include <functional>
#include <iostream>
#include <vector>

#include "colouring.hpp"
#include "hip/hip_runtime_api.h"
#include "kernels/res_calc.hpp"

template <bool SOA> Problem<SOA> initProblem(const std::string &input_dir) {
  std::ifstream mesh_res(input_dir + "mesh_res");
  std::ifstream mesh_x(input_dir + "mesh_x");
  std::ifstream mesh_q(input_dir + "mesh_q");
  std::ifstream mesh_adt(input_dir + "mesh_adt");
  Problem<SOA> problem(
      std::vector<std::istream *>{&mesh_res, &mesh_x, &mesh_q, &mesh_adt},
      std::vector<MY_SIZE>{res_calc::MAPPING_DIM, res_calc::MAPPING_DIM,
                           res_calc::MAPPING_DIM, res_calc::MAPPING_DIM},
      std::vector<std::pair<MY_SIZE, unsigned>>{
          {res_calc::RES_DIM, sizeof(double)},
          {res_calc::X_DIM, sizeof(double)},
          {res_calc::Q_DIM, sizeof(double)},
          {res_calc::ADT_DIM, sizeof(double)}},
      std::vector<std::pair<MY_SIZE, unsigned>>{});
  return problem;
}

template <bool SOA>
void readData(const std::string &input_dir, Problem<SOA> &problem) {
  std::ifstream data_res(input_dir + "data_res");
  std::ifstream data_x(input_dir + "data_x");
  std::ifstream data_q(input_dir + "data_q");
  std::ifstream data_adt(input_dir + "data_adt");
  problem.template readPointData<double>(data_res, 0);
  problem.template readPointData<double>(data_x, 1);
  problem.template readPointData<double>(data_q, 2);
  problem.template readPointData<double>(data_adt, 3);
}

template <bool SOA>
void writeData(const std::string &output_file, const Problem<SOA> &problem) {
  std::ofstream f(output_file);
  problem.template writePointData<double>(f);
}

template <bool SOA>
void runProblem(const std::string &input_dir, MY_SIZE num,
                const std::string &output_dir) {
  Problem<SOA> problem = initProblem<SOA>(input_dir + "/");
  std::string fname_base = output_dir + "/out_" + (SOA ? "SOA" : "AOS") + "_";

  readData(input_dir + "/", problem);
  problem.template loopCPUCellCentred<res_calc::StepSeq>(num);
  writeData(fname_base + "seq", problem);

  readData(input_dir + "/", problem);
  problem.template loopCPUCellCentredOMP<res_calc::StepOMP>(num);
  writeData(fname_base + "omp", problem);

  readData(input_dir + "/", problem);
  problem.template loopGPUCellCentred<res_calc::StepGPUGlobal>(num);
  writeData(fname_base + "glob", problem);

  readData(input_dir + "/", problem);
  problem.template loopGPUHierarchical<res_calc::StepGPUHierarchical>(num);
  writeData(fname_base + "hier", problem);
}

void runProblem(const std::string &input_dir, MY_SIZE num,
                const std::string &output_dir) {
  runProblem<false>(input_dir, num, output_dir);
  runProblem<true>(input_dir, num, output_dir);
}

template <bool SOA>
void testReordering(const std::string &input_dir, MY_SIZE num, bool partition) {
  std::cout << "========================================" << std::endl;
  std::cout << "Airfoil reordering test ";
  std::cout << (SOA ? "SOA" : "AOS");
  std::cout << std::endl << "Iteration: " << num;
  std::cout << " Partition: " << std::boolalpha << partition;
  std::cout << std::endl;
  std::cout << "========================================" << std::endl;
  Problem<SOA> problem1 = initProblem<SOA>(input_dir + "/");
  readData(input_dir + "/", problem1);
  Problem<SOA> problem2 = initProblem<SOA>(input_dir + "/");
  readData(input_dir + "/", problem2);

  problem1.reorder();
  if (partition) {
    problem1.partition(1.001);
    problem1.reorderToPartition();
    problem1.renumberPoints();
  }

  problem1.template loopGPUHierarchical<res_calc::StepGPUHierarchical>(num);
  problem2.template loopGPUHierarchical<res_calc::StepGPUHierarchical>(num);

  double max_diff = 0;
  const MY_SIZE num_points = problem1.mesh.numPoints(0);
  for (MY_SIZE i = 0; i < num_points; ++i) {
    for (unsigned d = 0; d < res_calc::RES_DIM; ++d) {
      const MY_SIZE ind1 = index<SOA>(
          num_points, problem1.applied_permutation[i], res_calc::RES_DIM, d);
      const MY_SIZE ind2 = index<SOA>(num_points, i, res_calc::RES_DIM, d);
      const double data1 =
          problem1.point_weights[0].template operator[]<double>(ind1);
      const double data2 =
          problem2.point_weights[0].template operator[]<double>(ind2);
      const double diff = std::abs(data1 - data2) /
                          (std::min(std::abs(data1), std::abs(data2)) + 1e-6);
      if (max_diff < diff) {
        max_diff = diff;
      }
    }
  }

  std::cout << "Test considered " << (max_diff < 1e-5 ? "PASSED" : "FAILED")
            << std::endl;
}

void testReordering(const std::string &input_dir, MY_SIZE num) {
  testReordering<false>(input_dir, num, false);
  testReordering<false>(input_dir, num, true);
  testReordering<true>(input_dir, num, false);
  testReordering<true>(input_dir, num, true);
}

template <bool SOA>
void measurement(const std::string &input_dir, MY_SIZE num) {

  {
    std::cout << "Running non reordered" << std::endl;
    Problem<SOA> problem = initProblem<SOA>(input_dir + "/");
    readData(input_dir + "/", problem);
    std::cout << "Data read." << std::endl;
    problem.template loopGPUHierarchical<res_calc::StepGPUHierarchical>(num);
  }

  {
    std::cout << "Running GPS reordered" << std::endl;
    Problem<SOA> problem = initProblem<SOA>(input_dir + "/");
    readData(input_dir + "/", problem);
    TIMER_START(timer_gps);
    problem.reorder();
    TIMER_PRINT(timer_gps, "reordering");
    problem.template loopGPUHierarchical<res_calc::StepGPUHierarchical>(num);
  }

  {
    std::cout << "Running partitioned" << std::endl;
    Problem<SOA> problem = initProblem<SOA>(input_dir + "/");
    readData(input_dir + "/", problem);
    TIMER_START(timer_metis);
    problem.reorder();
    problem.partition(1.001);
    problem.reorderToPartition();
    problem.renumberPoints();
    TIMER_PRINT(timer_metis, "partitioning");
    problem.template loopGPUHierarchical<res_calc::StepGPUHierarchical>(num);
  }
}

void measurement(const std::string &input_dir, MY_SIZE num) {
  std::cout << "AOS" << std::endl;
  measurement<false>(input_dir, num);
  std::cout << "SOA" << std::endl;
  measurement<true>(input_dir, num);
}

void printUsageTest(const char *program_name) {
  std::cerr << "Usage: " << program_name
            << " <input_dir> <output_dir> <iteration_number>" << std::endl;
}

void printUsageMeasure(const char *program_name) {
  std::cerr << "Usage: " << program_name << " <input_dir> <iteration_number>"
            << std::endl;
}

int mainMeasure(int argc, char *argv[]) {
  if (argc < 3) {
    printUsageMeasure(argv[0]);
    return 1;
  }
  measurement(argv[1], std::atol(argv[2]));
  return 0;
}

int mainTest(int argc, char *argv[]) {
  if (argc < 4) {
    printUsageTest(argv[0]);
    return 1;
  }
  runProblem(argv[1], std::atol(argv[3]), argv[2]);
  testReordering(argv[1], std::atol(argv[3]));
  return 0;
}

int main(int argc, char *argv[]) { return mainTest(argc, argv); }
